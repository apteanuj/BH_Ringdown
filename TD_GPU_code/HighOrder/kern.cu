#include "hip/hip_runtime.h"
#define Power pow
#define Cos cos
#define Sin sin
#define Cot 1.0/tan
#define Csc 1.0/sin

#include "parm.h"
#include "mycmplx.h"

__device__ double qre[M * N];
__device__ double qim[M * N];
__device__ double pre[M * N];
__device__ double pim[M * N];

__device__ double tred[M][N];
__device__ double timd[M][N];

__device__ double qre_c[M * N];
__device__ double qim_c[M * N];
__device__ double pre_c[M * N];
__device__ double pim_c[M * N];

__device__ double bb_c[M * N];
__device__ double cc_c[M * N];
__device__ double ee_c[M * N];
__device__ double a_re_c[M * N];
__device__ double a_im_c[M * N];
__device__ double d_re_c[M * N];
__device__ double d_im_c[M * N];
__device__ double v_re_c[M * N];
__device__ double v_im_c[M * N];

__device__ double rhs_qre[M * N];
__device__ double rhs_qim[M * N];
__device__ double rhs_pre[M * N];
__device__ double rhs_pim[M * N];

__device__ double mass;
__device__ double dtheta;
__device__ double dx;
__device__ double dt;
__device__ double aa;
__device__ double ss;
__device__ double mm;

__device__ double third = 0.33333333333333333;

__device__ double wenoL (double f, double fm, double fp, double fm2, double fp2){

       double beta0, beta1, beta2, w0, w1, w2, ww0, ww1, ww2, eps, res;

         ww0 = 0.1;
         ww1 = 0.6;
         ww2 = 0.3;

         res = (ww0*(2.0*fm2-7.0*fm+11.0*f) + 
               ww1*(-1.0*fm+5.0*f+2.0*fp) + 
               ww2*(2.0*f+5.0*fp-1.0*fp2))*0.5*third;

         return (res);

}

__device__ double wenoR (double f, double fm, double fp, double fm2, double fp2){

       double beta0, beta1, beta2, w0, w1, w2, ww0, ww1, ww2, eps, res;

         ww0 = 0.1;         
         ww1 = 0.6;
         ww2 = 0.3;

         res = (ww0*(2.0*fp2-7.0*fp+11.0*f) + 
               ww1*(-1.0*fp+5.0*f+2.0*fm) + 
               ww2*(2.0*f+5.0*fm-1.0*fm2))*0.5*third;

         return (res);

}



/* -------------------------------------------------- */

__global__ void
kernel_init (double dtheta_in, double mass_in, double aa_in, double ss_in, double mm_in,
	     double dx_in, double dt_in, double *theta, double *r_c)
{
  int k;
  int j;

  double rpt, ypt, stheta, spin, Om, Omstr, coefL, Omstrstr;
  double eR, bir,  Ataurho, Arhorho, Ayy, BtauReal; 
  double BtauImag, BrhoReal, By, BrhoImag, CReal, CImag;  
  double dbdx, em, rin, Rrpt;

  k = blockIdx.y * blockDim.y + threadIdx.y;
  j = blockIdx.x * blockDim.x + threadIdx.x;

  mass = mass_in;
  aa = aa_in;
  ss = ss_in;
  mm = mm_in;
  dtheta = dtheta_in;
  dx = dx_in;
  dt = dt_in;

  spin = aa;
  em = mm;

  rin = Xmin + (Xmax - Xmin) * 0.64;
  Rrpt = Xmax + 1e-6;

  if (k >= 3 && k < M - 3)
      {

         ypt = theta[k];
         rpt = Xmin + (double) j *dx;

         if (rpt<rin) {
         Om = 1.0;
         Omstr = 0.0;
         Omstrstr = 0.0;
         } else {
         Om = 1.0 - pow((rpt-rin)/(Rrpt-rin),4); 
         Omstr = -4.0*pow((rpt-rin),3)/pow((Rrpt-rin),4);
         Omstrstr = -12.0*pow((rpt-rin),2)/pow((Rrpt-rin),4);
         }

         coefL    = Om - rpt*Omstr;
         eR = Om*r_c[j];

         bir = -(((2.0*coefL - Power(Om,2))*Power(Power(eR,2)+Power(Om,2)*Power(spin,2),2))/Power(coefL,2))
            + Power(spin,2)*(Power(eR,2) - 2.0*eR*mass*Om + Power(Om,2)*Power(spin,2))*Power(Sin(ypt),2);

         Ataurho = (2.0*(coefL - Power(Om,2))*Power(Power(eR,2) + Power(Om,2)*Power(spin,2),2))/
            (bir*Power(coefL,2));
         Arhorho = -((Power(Om,2)*Power(Power(eR,2) + Power(Om,2)*Power(spin,2),2))/(bir*Power(coefL,2)));
         Ayy = -((Power(eR,2) - 2.0*eR*mass*Om + Power(Om,2)*Power(spin,2))/bir);
         BtauReal = -((Om*(Power(eR,3)*(4.0*Power(coefL,2)*(eR - 3.0*mass*Om) + 2.0*coefL*Power(eR,2)*Omstr + Power(eR,2)*Om*Omstrstr*rpt) + 2.0*eR*(Power(coefL,2)*Power(Om,2)*(5.0*eR - 4.0*mass*Om) - 3.0*Power(coefL,3)*(eR - 2.0*mass*Om) + 2.0*coefL*Power(eR,2)*Power(Om,2)*Omstr + Power(eR,2)*Power(Om,3)*Omstrstr*rpt)*Power(spin,2) + (-6.0*Power(coefL,3)*Power(Om,2) + 6.0*Power(coefL,2)*Power(Om,4) + 2.0*coefL*eR*Power(Om,4)*Omstr + eR*Power(Om,5)*Omstrstr*rpt)*Power(spin,4)))/(bir*Power(coefL,3)*eR));
         BrhoReal = -((Om*(Power(eR,5)*(2.0*Om*Omstr - 2.0*Power(Omstr,2)*rpt + Om*Omstrstr*rpt) + 2.0*Power(eR,3)*Power(Om,2)*(2.0*Om*Omstr - 2.0*Power(Omstr,2)*rpt + Om*Omstrstr*rpt)*Power(spin,2) + Power(Om,4)*(6.0*Power(Om,2) + 2.0*Power(Omstr,2)*rpt*(-eR + 3.0*rpt) + Om*(2.0*eR*Omstr - 12.0*Omstr*rpt + eR*Omstrstr*rpt))*Power(spin,4) + 2.0*Power(coefL,2)*eR*(2*Power(eR,3) - 6.0*Power(eR,2)*mass*Om + 5.0*eR*Power(Om,2)*Power(spin,2) - 4.0*mass*Power(Om,3)*Power(spin,2))))/(bir*Power(coefL,3)*eR));
         By = -(((Power(eR,2) - 2.0*eR*mass*Om + Power(Om,2)*Power(spin,2))*Cot(ypt))/bir);
         CReal = ((Power(eR,2) - 2.0*eR*mass*Om + Power(Om,2)*Power(spin,2))*(2.0*Power(eR,2) + 6.0*eR*mass*Om - 6.0*Power(Om,2)*Power(spin,2) + 4.0*Power(eR,2)*Power(Cot(ypt),2) - 4.0*em*Power(eR,2)*Cot(ypt)*Csc(ypt) + Power(em,2)*Power(eR,2)*Power(Csc(ypt),2)))/(bir*Power(eR,2));
         BtauImag = (2.0*spin*(em*(-(Power(Om,2)*(Power(eR,2) + Power(Om,2)*Power(spin,2))) + coefL*(Power(eR,2) + 2.0*eR*mass*Om + Power(Om,2)*Power(spin,2))) - 2.0*coefL*(Power(eR,2) - 2.0*eR*mass*Om + Power(Om,2)*Power(spin,2))*Cos(ypt)))/(bir*coefL);
         BrhoImag = (-2.0*em*Power(Om,2)*spin*(Power(eR,2) + Power(Om,2)*Power(spin,2)))/(bir*coefL);
         CImag = (2.0*em*Om*spin*(Power(eR,2) + 2.0*eR*mass*Om - 3.0*Power(Om,2)*Power(spin,2)))/(bir*eR);

         bb_c[idx (k, j)] = - (Ataurho + sqrt(Ataurho*Ataurho+4.0*Arhorho))/2.0;
         cc_c[idx (k, j)] = Ayy;
         ee_c[idx (k, j)] = Ataurho + bb_c[idx (k, j)];

         a_re_c[idx (k, j)] = -BtauReal;
         a_im_c[idx (k, j)] = -BtauImag;

         d_re_c[idx (k, j)] = BrhoReal -bb_c[idx (k, j)]*BtauReal; 
         d_im_c[idx (k, j)] = BrhoImag -bb_c[idx (k, j)]*BtauImag;

         v_re_c[idx (k, j)] = -CReal;
         v_im_c[idx (k, j)] = -CImag;


        }

        
         qre[idx (k, j)] = 0.0;
         qim[idx (k, j)] = 0.0;
         pre[idx (k, j)] = 0.0;
         pim[idx (k, j)] = 0.0;

}

__global__ void
kernel_rhs1(double *theta, double *r_c)
{
  int b;
  int a;

  b = blockIdx.y * blockDim.y + threadIdx.y;
  a = blockIdx.x * blockDim.x + threadIdx.x;

  double ctan, stheta, ctheta, dbdx, dre;
  double qrey = 0.0;
  double qreyy = 0.0;
  double qimy = 0.0;
  double qimyy = 0.0;
  double ll_re, ll_im, f_mid_p, f_mid_m;
  double over_dx, over_dtheta, over_dt;

         over_dx = 1.0 / dx;
         over_dtheta = 1.0 / dtheta;
         over_dt = 1.0 / dt;

         dbdx = 0.0;
         if (a >= 3 && a < N-3)
         dbdx = (45.0*bb_c[idx (b, a+1)]-45.0*bb_c[idx (b, a-1)]
                  -9.0*bb_c[idx (b, a+2)]+9.0*bb_c[idx (b, a-2)] 
                  +bb_c[idx (b, a+3)]-bb_c[idx (b, a-3)])*(0.016666666666667*over_dx);
          dre = d_re_c[idx (b, a)] - dbdx*ee_c[idx (b, a)];


  if (b >= 3 && b < M - 3)
    if (a >= 3 && a < N - 3)
      {

	ctan = tan (0.5 * Pi - theta[b]);					

        qrey = ( (qre[idx (b + 3, a)]-qre[idx (b - 3, a)])
            -9.0*(qre[idx (b + 2, a)]-qre[idx (b - 2, a)])
            +45.0*(qre[idx (b + 1, a)]-qre[idx (b - 1, a)]) ) * (0.05*third*over_dtheta);

        qreyy = ( 2.0*(qre[idx (b + 3, a)]+qre[idx (b - 3, a)])
            -27.0*(qre[idx (b + 2, a)]+qre[idx (b - 2, a)])
            +270.0*(qre[idx (b + 1, a)]+qre[idx (b - 1, a)]) 
            -490.0*qre[idx (b, a)] ) * (0.05*third*third*over_dtheta*over_dtheta);

        qimy = ( (qim[idx (b + 3, a)]-qim[idx (b - 3, a)])
            -9.0*(qim[idx (b + 2, a)]-qim[idx (b - 2, a)])
            +45.0*(qim[idx (b + 1, a)]-qim[idx (b - 1, a)]) ) * (0.05*third*over_dtheta);

        qimyy = ( 2.0*(qim[idx (b + 3, a)]+qim[idx (b - 3, a)])
            -27.0*(qim[idx (b + 2, a)]+qim[idx (b - 2, a)])
            +270.0*(qim[idx (b + 1, a)]+qim[idx (b - 1, a)]) 
            -490.0*qim[idx (b, a)] ) * (0.05*third*third*over_dtheta*over_dtheta);


	ll_re = qreyy + ctan * qrey;
	ll_im = qimyy + ctan * qimy;

        double  qredss = 0.0;
        double  qimdss = 0.0;
        double  predss = 0.0;
        double  pimdss = 0.0;

	double qrex = (45.0*qre[idx (b, a+1)]-45.0*qre[idx (b, a-1)]
                       -9.0*qre[idx (b, a+2)]+9.0*qre[idx (b, a-2)]
                       +qre[idx (b, a+3)]-qre[idx (b, a-3)])*(0.016666666666667*over_dx);
	double qimx = (45.0*qim[idx (b, a+1)]-45.0*qim[idx (b, a-1)]
                       -9.0*qim[idx (b, a+2)]+9.0*qim[idx (b, a-2)]
                       +qim[idx (b, a+3)]-qim[idx (b, a-3)])*(0.016666666666667*over_dx);
	double prex = (45.0*pre[idx (b, a+1)]-45.0*pre[idx (b, a-1)]
                       -9.0*pre[idx (b, a+2)]+9.0*pre[idx (b, a-2)]
                       +pre[idx (b, a+3)]-pre[idx (b, a-3)])*(0.016666666666667*over_dx);
	double pimx = (45.0*pim[idx (b, a+1)]-45.0*pim[idx (b, a-1)]
                       -9.0*pim[idx (b, a+2)]+9.0*pim[idx (b, a-2)]
                       +pim[idx (b, a+3)]-pim[idx (b, a-3)])*(0.016666666666667*over_dx);


         if ((a >= 4)&&(a < N-4)) {
         qredss = qre[idx (b,a+4)]+qre[idx (b,a-4)] - 8.0*(qre[idx (b,a+3)]+qre[idx (b,a-3)]) + 70.0*qre[idx (b,a)]
           - 56.0*(qre[idx (b,a+1)]+qre[idx (b,a-1)]) + 28.0*(qre[idx (b,a+2)]+qre[idx (b,a-2)]);   
         qimdss = qim[idx (b,a+4)]+qim[idx (b,a-4)] - 8.0*(qim[idx (b,a+3)]+qim[idx (b,a-3)]) + 70.0*qim[idx (b,a)]
           - 56.0*(qim[idx (b,a+1)]+qim[idx (b,a-1)]) + 28.0*(qim[idx (b,a+2)]+qim[idx (b,a-2)]);   
         predss = pre[idx (b,a+4)]+pre[idx (b,a-4)] - 8.0*(pre[idx (b,a+3)]+pre[idx (b,a-3)]) + 70.0*pre[idx (b,a)]
           - 56.0*(pre[idx (b,a+1)]+pre[idx (b,a-1)]) + 28.0*(pre[idx (b,a+2)]+pre[idx (b,a-2)]);   
         pimdss = pim[idx (b,a+4)]+pim[idx (b,a-4)] - 8.0*(pim[idx (b,a+3)]+pim[idx (b,a-3)]) + 70.0*pim[idx (b,a)]
           - 56.0*(pim[idx (b,a+1)]+pim[idx (b,a-1)]) + 28.0*(pim[idx (b,a+2)]+pim[idx (b,a-2)]);   
         }

         qredss = 0.1 * qredss * 0.0625 * over_dt;
         qimdss = 0.1 * qimdss * 0.0625 * over_dt;
         predss = 0.1 * predss * 0.0625 * over_dt;
         pimdss = 0.1 * pimdss * 0.0625 * over_dt;

/*
        f_mid_p = wenoL(pre[idx (b,a)], pre[idx (b,a-1)], pre[idx (b,a+1)], pre[idx (b,a-2)], pre[idx (b,a+2)]); 
        f_mid_m = wenoL(pre[idx (b,a-1)], pre[idx (b,a-2)], pre[idx (b,a)], pre[idx (b,a-3)], pre[idx (b,a+1)]); 
        double  prex_wno = (f_mid_p - f_mid_m) * over_dx;

        f_mid_p = wenoL(pim[idx (b,a)], pim[idx (b,a-1)], pim[idx (b,a+1)], pim[idx (b,a-2)], pim[idx (b,a+2)]); 
        f_mid_m = wenoL(pim[idx (b,a-1)], pim[idx (b,a-2)], pim[idx (b,a)], pim[idx (b,a-3)], pim[idx (b,a+1)]); 
        double  pimx_wno = (f_mid_p - f_mid_m) * over_dx;

        f_mid_m = wenoR(qre[idx (b,a)], qre[idx (b,a-1)], qre[idx (b,a+1)], qre[idx (b,a-2)], qre[idx (b,a+2)]); 
        f_mid_p = wenoR(qre[idx (b,a+1)], qre[idx (b,a)], qre[idx (b,a+2)], qre[idx (b,a-1)], qre[idx (b,a+3)]); 
        double  qrex_wno = (f_mid_p - f_mid_m) * over_dx;

        f_mid_m = wenoR(qim[idx (b,a)], qim[idx (b,a-1)], qim[idx (b,a+1)], qim[idx (b,a-2)], qim[idx (b,a+2)]); 
        f_mid_p = wenoR(qim[idx (b,a+1)], qim[idx (b,a)], qim[idx (b,a+2)], qim[idx (b,a-1)], qim[idx (b,a+3)]); 
        double  qimx_wno = (f_mid_p - f_mid_m) * over_dx;
*/

	rhs_qre[idx (b, a)] = -bb_c[idx (b, a)] * qrex + pre[idx (b, a)] -qredss;
	rhs_qim[idx (b, a)] = -bb_c[idx (b, a)] * qimx + pim[idx (b, a)] -qimdss;

	rhs_pre[idx (b, a)] = tred[b][a] +
	  ee_c[idx (b, a)] * prex           -predss
	  + cc_c[idx (b, a)] * ll_re
	  + dre * qrex - d_im_c[idx (b, a)] * qimx
	  - a_re_c[idx (b, a)] * pre[idx (b, a)] + a_im_c[idx (b, a)] * pim[idx (b, a)]
	  - v_re_c[idx (b, a)] * qre[idx (b, a)] + v_im_c[idx (b, a)] * qim[idx (b, a)];

	rhs_pim[idx (b, a)] = timd[b][a] +
	  ee_c[idx (b, a)] * pimx           -pimdss
	  + cc_c[idx (b, a)] * ll_im
	  + dre * qimx + d_im_c[idx (b, a)] * qrex
	  - a_re_c[idx (b, a)] * pim[idx (b, a)] - a_im_c[idx (b, a)] * pre[idx (b, a)]
	  - v_re_c[idx (b, a)] * qim[idx (b, a)] - v_im_c[idx (b, a)] * qre[idx (b, a)];

      }
}

__global__ void
kernel_update1 ()
{
  int b;
  int a;

  b = blockIdx.y * blockDim.y + threadIdx.y;
  a = blockIdx.x * blockDim.x + threadIdx.x;

  if (b >= 3 && b < M - 3)
    if (a >= 3 && a < N - 3)
      {
	qre_c[idx (b, a)] = qre[idx (b, a)] + dt * rhs_qre[idx (b, a)];
	qim_c[idx (b, a)] = qim[idx (b, a)] + dt * rhs_qim[idx (b, a)];
	pre_c[idx (b, a)] = pre[idx (b, a)] + dt * rhs_pre[idx (b, a)];
	pim_c[idx (b, a)] = pim[idx (b, a)] + dt * rhs_pim[idx (b, a)];
      }
}

__global__ void
kernel_boundary1a ()
{
  int a;

  a = blockIdx.x * blockDim.x + threadIdx.x;

  if (a >= 3 && a < N - 3){

  if (((int)mm)%2==0) { 

  qre_c[idx (2, a)] = (4.0 * qre_c[idx (3, a)] - qre_c[idx (4, a)]) * third;
  qim_c[idx (2, a)] = (4.0 * qim_c[idx (3, a)] - qim_c[idx (4, a)]) * third;
  pre_c[idx (2, a)] = (4.0 * pre_c[idx (3, a)] - pre_c[idx (4, a)]) * third;
  pim_c[idx (2, a)] = (4.0 * pim_c[idx (3, a)] - pim_c[idx (4, a)]) * third;

  qre_c[idx (1, a)] = (4.0 * qre_c[idx (2, a)] - qre_c[idx (3, a)]) * third;
  qim_c[idx (1, a)] = (4.0 * qim_c[idx (2, a)] - qim_c[idx (3, a)]) * third;
  pre_c[idx (1, a)] = (4.0 * pre_c[idx (2, a)] - pre_c[idx (3, a)]) * third;
  pim_c[idx (1, a)] = (4.0 * pim_c[idx (2, a)] - pim_c[idx (3, a)]) * third;

  qre_c[idx (M - 3, a)] =
    (4.0 * qre_c[idx (M - 4, a)] - qre_c[idx (M - 5, a)]) * third;
  qim_c[idx (M - 3, a)] =
    (4.0 * qim_c[idx (M - 4, a)] - qim_c[idx (M - 5, a)]) * third;
  pre_c[idx (M - 3, a)] =
    (4.0 * pre_c[idx (M - 4, a)] - pre_c[idx (M - 5, a)]) * third;
  pim_c[idx (M - 3, a)] =
    (4.0 * pim_c[idx (M - 4, a)] - pim_c[idx (M - 5, a)]) * third;

  qre_c[idx (M - 2, a)] =
    (4.0 * qre_c[idx (M - 3, a)] - qre_c[idx (M - 4, a)]) * third;
  qim_c[idx (M - 2, a)] =
    (4.0 * qim_c[idx (M - 3, a)] - qim_c[idx (M - 4, a)]) * third;
  pre_c[idx (M - 2, a)] =
    (4.0 * pre_c[idx (M - 3, a)] - pre_c[idx (M - 4, a)]) * third;
  pim_c[idx (M - 2, a)] =
    (4.0 * pim_c[idx (M - 3, a)] - pim_c[idx (M - 4, a)]) * third;

  qre_c[idx (0, a)] = (4.0 * qre_c[idx (1, a)] - qre_c[idx (2, a)]) * third;
  qim_c[idx (0, a)] = (4.0 * qim_c[idx (1, a)] - qim_c[idx (2, a)]) * third;
  pre_c[idx (0, a)] = (4.0 * pre_c[idx (1, a)] - pre_c[idx (2, a)]) * third;
  pim_c[idx (0, a)] = (4.0 * pim_c[idx (1, a)] - pim_c[idx (2, a)]) * third;

  qre_c[idx (M - 1, a)] =
    (4.0 * qre_c[idx (M - 2, a)] - qre_c[idx (M - 3, a)]) * third;
  qim_c[idx (M - 1, a)] =
    (4.0 * qim_c[idx (M - 2, a)] - qim_c[idx (M - 3, a)]) * third;
  pre_c[idx (M - 1, a)] =
    (4.0 * pre_c[idx (M - 2, a)] - pre_c[idx (M - 3, a)]) * third;
  pim_c[idx (M - 1, a)] =
    (4.0 * pim_c[idx (M - 2, a)] - pim_c[idx (M - 3, a)]) * third;

  } else {

  qre_c[idx (0, a)] = 0.0;
  qim_c[idx (0, a)] = 0.0;
  pre_c[idx (0, a)] = 0.0;
  pim_c[idx (0, a)] = 0.0;

  qre_c[idx (M - 1, a)] = 0.0;
  qim_c[idx (M - 1, a)] = 0.0;
  pre_c[idx (M - 1, a)] = 0.0;
  pim_c[idx (M - 1, a)] = 0.0;

  qre_c[idx (1, a)] = 0.0;
  qim_c[idx (1, a)] = 0.0;
  pre_c[idx (1, a)] = 0.0;
  pim_c[idx (1, a)] = 0.0;

  qre_c[idx (2, a)] = 0.0;
  qim_c[idx (2, a)] = 0.0;
  pre_c[idx (2, a)] = 0.0;
  pim_c[idx (2, a)] = 0.0;

  qre_c[idx (M - 2, a)] = 0.0;
  qim_c[idx (M - 2, a)] = 0.0;
  pre_c[idx (M - 2, a)] = 0.0;
  pim_c[idx (M - 2, a)] = 0.0;

  qre_c[idx (M - 3, a)] = 0.0;
  qim_c[idx (M - 3, a)] = 0.0;
  pre_c[idx (M - 3, a)] = 0.0;
  pim_c[idx (M - 3, a)] = 0.0;

  }

 }

}


__global__ void
kernel_boundary1b ()
{
  int b;

  b = blockIdx.x * blockDim.x + threadIdx.x;

  qre_c[idx (b, 2)] = (4.0*qre_c[idx (b, 3)]-qre_c[idx (b, 4)]) * third;
  qim_c[idx (b, 2)] = (4.0*qim_c[idx (b, 3)]-qim_c[idx (b, 4)]) * third;
  qre_c[idx (b, N - 3)] = (4.0*qre_c[idx (b, N-4)]-qre_c[idx (b, N-5)]) * third;
  qim_c[idx (b, N - 3)] = (4.0*qim_c[idx (b, N-4)]-qim_c[idx (b, N-5)]) * third;

  qre_c[idx (b, 1)] = (4.0*qre_c[idx (b, 2)]-qre_c[idx (b, 3)]) * third;
  qim_c[idx (b, 1)] = (4.0*qim_c[idx (b, 2)]-qim_c[idx (b, 3)]) * third;
  qre_c[idx (b, N - 2)] = (4.0*qre_c[idx (b, N-3)]-qre_c[idx (b, N-4)]) * third;
  qim_c[idx (b, N - 2)] = (4.0*qim_c[idx (b, N-3)]-qim_c[idx (b, N-4)]) * third;

  qre_c[idx (b, 0)] = (4.0*qre_c[idx (b, 1)]-qre_c[idx (b, 2)]) * third;
  qim_c[idx (b, 0)] = (4.0*qim_c[idx (b, 1)]-qim_c[idx (b, 2)]) * third;
  qre_c[idx (b, N - 1)] = (4.0*qre_c[idx (b, N-2)]-qre_c[idx (b, N-3)]) * third;
  qim_c[idx (b, N - 1)] = (4.0*qim_c[idx (b, N-2)]-qim_c[idx (b, N-3)]) * third;

  pre_c[idx (b, 2)] = (4.0*pre_c[idx (b, 3)]-pre_c[idx (b, 4)]) * third;
  pim_c[idx (b, 2)] = (4.0*pim_c[idx (b, 3)]-pim_c[idx (b, 4)]) * third;
  pre_c[idx (b, N - 3)] = (4.0*pre_c[idx (b, N-4)]-pre_c[idx (b, N-5)]) * third;
  pim_c[idx (b, N - 3)] = (4.0*pim_c[idx (b, N-4)]-pim_c[idx (b, N-5)]) * third;

  pre_c[idx (b, 1)] = (4.0*pre_c[idx (b, 2)]-pre_c[idx (b, 3)]) * third;
  pim_c[idx (b, 1)] = (4.0*pim_c[idx (b, 2)]-pim_c[idx (b, 3)]) * third;
  pre_c[idx (b, N - 2)] = (4.0*pre_c[idx (b, N-3)]-pre_c[idx (b, N-4)]) * third;
  pim_c[idx (b, N - 2)] = (4.0*pim_c[idx (b, N-3)]-pim_c[idx (b, N-4)]) * third;

  pre_c[idx (b, 0)] = (4.0*pre_c[idx (b, 1)]-pre_c[idx (b, 2)]) * third;
  pim_c[idx (b, 0)] = (4.0*pim_c[idx (b, 1)]-pim_c[idx (b, 2)]) * third;
  pre_c[idx (b, N - 1)] = (4.0*pre_c[idx (b, N-2)]-pre_c[idx (b, N-3)]) * third;
  pim_c[idx (b, N - 1)] = (4.0*pim_c[idx (b, N-2)]-pim_c[idx (b, N-3)]) * third;

}


__global__ void
kernel_rhs2(double *theta, double *r_c)
{
  int b;
  int a;

  b = blockIdx.y * blockDim.y + threadIdx.y;
  a = blockIdx.x * blockDim.x + threadIdx.x;

  double ctan, stheta, ctheta, dbdx, dre;
  double qrey = 0.0;
  double qreyy = 0.0;
  double qimy = 0.0;
  double qimyy = 0.0;
  double ll_re, ll_im, f_mid_p, f_mid_m;
  double over_dx, over_dtheta, over_dt;

         over_dx = 1.0 / dx;
         over_dtheta = 1.0 / dtheta;
         over_dt = 1.0 / dt;

         dbdx = 0.0;
         if (a >= 3 && a < N-3)
         dbdx = (45.0*bb_c[idx (b, a+1)]-45.0*bb_c[idx (b, a-1)]
                  -9.0*bb_c[idx (b, a+2)]+9.0*bb_c[idx (b, a-2)] 
                  +bb_c[idx (b, a+3)]-bb_c[idx (b, a-3)])*(0.016666666666667*over_dx);
          dre = d_re_c[idx (b, a)] - dbdx*ee_c[idx (b, a)];


  if (b >= 3 && b < M - 3)
    if (a >= 3 && a < N - 3)
      {

	ctan = tan (0.5 * Pi - theta[b]);

        qrey = ( (qre_c[idx (b + 3, a)]-qre_c[idx (b - 3, a)])
            -9.0*(qre_c[idx (b + 2, a)]-qre_c[idx (b - 2, a)])
            +45.0*(qre_c[idx (b + 1, a)]-qre_c[idx (b - 1, a)]) ) * (0.05*third*over_dtheta);

        qreyy = ( 2.0*(qre_c[idx (b + 3, a)]+qre_c[idx (b - 3, a)])
            -27.0*(qre_c[idx (b + 2, a)]+qre_c[idx (b - 2, a)])
            +270.0*(qre_c[idx (b + 1, a)]+qre_c[idx (b - 1, a)]) 
            -490.0*qre_c[idx (b, a)] ) * (0.05*third*third*over_dtheta*over_dtheta);

        qimy = ( (qim_c[idx (b + 3, a)]-qim_c[idx (b - 3, a)])
            -9.0*(qim_c[idx (b + 2, a)]-qim_c[idx (b - 2, a)])
            +45.0*(qim_c[idx (b + 1, a)]-qim_c[idx (b - 1, a)]) ) * (0.05*third*over_dtheta);

        qimyy = ( 2.0*(qim_c[idx (b + 3, a)]+qim_c[idx (b - 3, a)])
            -27.0*(qim_c[idx (b + 2, a)]+qim_c[idx (b - 2, a)])
            +270.0*(qim_c[idx (b + 1, a)]+qim_c[idx (b - 1, a)]) 
            -490.0*qim_c[idx (b, a)] ) * (0.05*third*third*over_dtheta*over_dtheta);

	ll_re = qreyy + ctan * qrey;
	ll_im = qimyy + ctan * qimy;

       double  qredss = 0.0;
       double  qimdss = 0.0;
       double  predss = 0.0;
       double  pimdss = 0.0;

	double qrex = (45.0*qre_c[idx (b, a+1)]-45.0*qre_c[idx (b, a-1)]
                       -9.0*qre_c[idx (b, a+2)]+9.0*qre_c[idx (b, a-2)]
                       +qre_c[idx (b, a+3)]-qre_c[idx (b, a-3)])*(0.016666666666667*over_dx);
	double qimx = (45.0*qim_c[idx (b, a+1)]-45.0*qim_c[idx (b, a-1)]
                       -9.0*qim_c[idx (b, a+2)]+9.0*qim_c[idx (b, a-2)]
                       +qim_c[idx (b, a+3)]-qim_c[idx (b, a-3)])*(0.016666666666667*over_dx);
	double prex = (45.0*pre_c[idx (b, a+1)]-45.0*pre_c[idx (b, a-1)]
                       -9.0*pre_c[idx (b, a+2)]+9.0*pre_c[idx (b, a-2)]
                       +pre_c[idx (b, a+3)]-pre_c[idx (b, a-3)])*(0.016666666666667*over_dx);
	double pimx = (45.0*pim_c[idx (b, a+1)]-45.0*pim_c[idx (b, a-1)]
                       -9.0*pim_c[idx (b, a+2)]+9.0*pim_c[idx (b, a-2)]
                       +pim_c[idx (b, a+3)]-pim_c[idx (b, a-3)])*(0.016666666666667*over_dx);


         if ((a >= 4)&&(a < N-4)) {
         qredss = qre_c[idx (b,a+4)]+qre_c[idx (b,a-4)] - 8.0*(qre_c[idx (b,a+3)]+qre_c[idx (b,a-3)]) + 70.0*qre_c[idx (b,a)]
           - 56.0*(qre_c[idx (b,a+1)]+qre_c[idx (b,a-1)]) + 28.0*(qre_c[idx (b,a+2)]+qre_c[idx (b,a-2)]);   
         qimdss = qim_c[idx (b,a+4)]+qim_c[idx (b,a-4)] - 8.0*(qim_c[idx (b,a+3)]+qim_c[idx (b,a-3)]) + 70.0*qim_c[idx (b,a)]
           - 56.0*(qim_c[idx (b,a+1)]+qim_c[idx (b,a-1)]) + 28.0*(qim_c[idx (b,a+2)]+qim_c[idx (b,a-2)]);   
         predss = pre_c[idx (b,a+4)]+pre_c[idx (b,a-4)] - 8.0*(pre_c[idx (b,a+3)]+pre_c[idx (b,a-3)]) + 70.0*pre_c[idx (b,a)]
           - 56.0*(pre_c[idx (b,a+1)]+pre_c[idx (b,a-1)]) + 28.0*(pre_c[idx (b,a+2)]+pre_c[idx (b,a-2)]);   
         pimdss = pim_c[idx (b,a+4)]+pim_c[idx (b,a-4)] - 8.0*(pim_c[idx (b,a+3)]+pim_c[idx (b,a-3)]) + 70.0*pim_c[idx (b,a)]
           - 56.0*(pim_c[idx (b,a+1)]+pim_c[idx (b,a-1)]) + 28.0*(pim_c[idx (b,a+2)]+pim_c[idx (b,a-2)]);   
         }

         qredss = 0.1 * qredss * 0.0625 * over_dt;
         qimdss = 0.1 * qimdss * 0.0625 * over_dt;
         predss = 0.1 * predss * 0.0625 * over_dt;
         pimdss = 0.1 * pimdss * 0.0625 * over_dt;

/*
        f_mid_p = wenoL(pre_c[idx (b,a)], pre_c[idx (b,a-1)], pre_c[idx (b,a+1)], pre_c[idx (b,a-2)], pre_c[idx (b,a+2)]); 
        f_mid_m = wenoL(pre_c[idx (b,a-1)], pre_c[idx (b,a-2)], pre_c[idx (b,a)], pre_c[idx (b,a-3)], pre_c[idx (b,a+1)]); 
        double  prex_wno = (f_mid_p - f_mid_m) * over_dx;

        f_mid_p = wenoL(pim_c[idx (b,a)], pim_c[idx (b,a-1)], pim_c[idx (b,a+1)], pim_c[idx (b,a-2)], pim_c[idx (b,a+2)]); 
        f_mid_m = wenoL(pim_c[idx (b,a-1)], pim_c[idx (b,a-2)], pim_c[idx (b,a)], pim_c[idx (b,a-3)], pim_c[idx (b,a+1)]); 
        double  pimx_wno = (f_mid_p - f_mid_m) * over_dx;

        f_mid_m = wenoR(qre_c[idx (b,a)], qre_c[idx (b,a-1)], qre_c[idx (b,a+1)], qre_c[idx (b,a-2)], qre_c[idx (b,a+2)]); 
        f_mid_p = wenoR(qre_c[idx (b,a+1)], qre_c[idx (b,a)], qre_c[idx (b,a+2)], qre_c[idx (b,a-1)], qre_c[idx (b,a+3)]); 
        double  qrex_wno = (f_mid_p - f_mid_m) * over_dx;

        f_mid_m = wenoR(qim_c[idx (b,a)], qim_c[idx (b,a-1)], qim_c[idx (b,a+1)], qim_c[idx (b,a-2)], qim_c[idx (b,a+2)]); 
        f_mid_p = wenoR(qim_c[idx (b,a+1)], qim_c[idx (b,a)], qim_c[idx (b,a+2)], qim_c[idx (b,a-1)], qim_c[idx (b,a+3)]); 
        double  qimx_wno = (f_mid_p - f_mid_m) * over_dx;
*/

	rhs_qre[idx (b, a)] = -bb_c[idx (b, a)] * qrex + pre_c[idx (b, a)] -qredss;
	rhs_qim[idx (b, a)] = -bb_c[idx (b, a)] * qimx + pim_c[idx (b, a)] -qimdss;

	rhs_pre[idx (b, a)] = tred[b][a] +
	  ee_c[idx (b, a)] * prex        -predss
	  + cc_c[idx (b, a)] * ll_re
	  + dre * qrex - d_im_c[idx (b, a)] * qimx
	  - a_re_c[idx (b, a)] * pre_c[idx (b, a)] + a_im_c[idx (b, a)] * pim_c[idx (b, a)]
	  - v_re_c[idx (b, a)] * qre_c[idx (b, a)] + v_im_c[idx (b, a)] * qim_c[idx (b, a)];


	rhs_pim[idx (b, a)] = timd[b][a] +
	  ee_c[idx (b, a)] * pimx        -pimdss
	  + cc_c[idx (b, a)] * ll_im
	  + dre * qimx + d_im_c[idx (b, a)] * qrex
	  - a_re_c[idx (b, a)] * pim_c[idx (b, a)] - a_im_c[idx (b, a)] * pre_c[idx (b, a)]
	  - v_re_c[idx (b, a)] * qim_c[idx (b, a)] - v_im_c[idx (b, a)] * qre_c[idx (b, a)];

      }
}

__global__ void
kernel_update2 ()
{
  int b;
  int a;

  b = blockIdx.y * blockDim.y + threadIdx.y;
  a = blockIdx.x * blockDim.x + threadIdx.x;

  if (b >= 3 && b < M - 3)
    if (a >= 3 && a < N - 3)
      {
	qre_c[idx (b, a)] = 0.25 * (3.0*qre[idx (b, a)] + qre_c[idx (b, a)] + dt * rhs_qre[idx (b, a)]);
	qim_c[idx (b, a)] = 0.25 * (3.0*qim[idx (b, a)] + qim_c[idx (b, a)] + dt * rhs_qim[idx (b, a)]);
	pre_c[idx (b, a)] = 0.25 * (3.0*pre[idx (b, a)] + pre_c[idx (b, a)] + dt * rhs_pre[idx (b, a)]);
	pim_c[idx (b, a)] = 0.25 * (3.0*pim[idx (b, a)] + pim_c[idx (b, a)] + dt * rhs_pim[idx (b, a)]);
      }

}

__global__ void
kernel_update3 ()
{
  int b;
  int a;

  b = blockIdx.y * blockDim.y + threadIdx.y;
  a = blockIdx.x * blockDim.x + threadIdx.x;

  if (b >= 3 && b < M - 3)
    if (a >= 3 && a < N - 3)
      {
	qre[idx (b, a)] = third * (qre[idx (b, a)] + 2.0*qre_c[idx (b, a)] + 2.0*dt * rhs_qre[idx (b, a)]);
	qim[idx (b, a)] = third * (qim[idx (b, a)] + 2.0*qim_c[idx (b, a)] + 2.0*dt * rhs_qim[idx (b, a)]);
	pre[idx (b, a)] = third * (pre[idx (b, a)] + 2.0*pre_c[idx (b, a)] + 2.0*dt * rhs_pre[idx (b, a)]);
	pim[idx (b, a)] = third * (pim[idx (b, a)] + 2.0*pim_c[idx (b, a)] + 2.0*dt * rhs_pim[idx (b, a)]);
      }

}

__global__ void
kernel_boundary2a ()
{
  int a;

  a = blockIdx.x * blockDim.x + threadIdx.x;

  if (a >= 3 && a < N - 3){

  if (((int)mm)%2==0) {

  qre[idx (2, a)] = (4.0 * qre[idx (3, a)] - qre[idx (4, a)]) * third;
  qim[idx (2, a)] = (4.0 * qim[idx (3, a)] - qim[idx (4, a)]) * third;
  pre[idx (2, a)] = (4.0 * pre[idx (3, a)] - pre[idx (4, a)]) * third;
  pim[idx (2, a)] = (4.0 * pim[idx (3, a)] - pim[idx (4, a)]) * third;

  qre[idx (1, a)] = (4.0 * qre[idx (2, a)] - qre[idx (3, a)]) * third;
  qim[idx (1, a)] = (4.0 * qim[idx (2, a)] - qim[idx (3, a)]) * third;
  pre[idx (1, a)] = (4.0 * pre[idx (2, a)] - pre[idx (3, a)]) * third;
  pim[idx (1, a)] = (4.0 * pim[idx (2, a)] - pim[idx (3, a)]) * third;

  qre[idx (M - 3, a)] =
    (4.0 * qre[idx (M - 4, a)] - qre[idx (M - 5, a)]) * third;
  qim[idx (M - 3, a)] =
    (4.0 * qim[idx (M - 4, a)] - qim[idx (M - 5, a)]) * third;
  pre[idx (M - 3, a)] =
    (4.0 * pre[idx (M - 4, a)] - pre[idx (M - 5, a)]) * third;
  pim[idx (M - 3, a)] =
    (4.0 * pim[idx (M - 4, a)] - pim[idx (M - 5, a)]) * third;

  qre[idx (M - 2, a)] =
    (4.0 * qre[idx (M - 3, a)] - qre[idx (M - 4, a)]) * third;
  qim[idx (M - 2, a)] =
    (4.0 * qim[idx (M - 3, a)] - qim[idx (M - 4, a)]) * third;
  pre[idx (M - 2, a)] =
    (4.0 * pre[idx (M - 3, a)] - pre[idx (M - 4, a)]) * third;
  pim[idx (M - 2, a)] =
    (4.0 * pim[idx (M - 3, a)] - pim[idx (M - 4, a)]) * third;

  qre[idx (0, a)] = (4.0 * qre[idx (1, a)] - qre[idx (2, a)]) * third;
  qim[idx (0, a)] = (4.0 * qim[idx (1, a)] - qim[idx (2, a)]) * third;
  pre[idx (0, a)] = (4.0 * pre[idx (1, a)] - pre[idx (2, a)]) * third;
  pim[idx (0, a)] = (4.0 * pim[idx (1, a)] - pim[idx (2, a)]) * third;

  qre[idx (M - 1, a)] =
    (4.0 * qre[idx (M - 2, a)] - qre[idx (M - 3, a)]) * third;
  qim[idx (M - 1, a)] =
    (4.0 * qim[idx (M - 2, a)] - qim[idx (M - 3, a)]) * third;
  pre[idx (M - 1, a)] =
    (4.0 * pre[idx (M - 2, a)] - pre[idx (M - 3, a)]) * third;
  pim[idx (M - 1, a)] =
    (4.0 * pim[idx (M - 2, a)] - pim[idx (M - 3, a)]) * third;

  } else {

  qre[idx (0, a)] = 0.0;
  qim[idx (0, a)] = 0.0;
  pre[idx (0, a)] = 0.0;
  pim[idx (0, a)] = 0.0;

  qre[idx (M - 1, a)] = 0.0;
  qim[idx (M - 1, a)] = 0.0;
  pre[idx (M - 1, a)] = 0.0;
  pim[idx (M - 1, a)] = 0.0;

  qre[idx (1, a)] = 0.0;
  qim[idx (1, a)] = 0.0;
  pre[idx (1, a)] = 0.0;
  pim[idx (1, a)] = 0.0;

  qre[idx (2, a)] = 0.0;
  qim[idx (2, a)] = 0.0;
  pre[idx (2, a)] = 0.0;
  pim[idx (2, a)] = 0.0;

  qre[idx (M - 2, a)] = 0.0;
  qim[idx (M - 2, a)] = 0.0;
  pre[idx (M - 2, a)] = 0.0;
  pim[idx (M - 2, a)] = 0.0;

  qre[idx (M - 3, a)] = 0.0;
  qim[idx (M - 3, a)] = 0.0;
  pre[idx (M - 3, a)] = 0.0;
  pim[idx (M - 3, a)] = 0.0;

  }

 }

}

__global__ void
kernel_boundary2b (double *d_qre_buff, double *d_qim_buff)
{
  int b, l, i;

  b = blockIdx.x * blockDim.x + threadIdx.x;

  qre[idx (b, 2)] = (4.0*qre[idx (b, 3)]-qre[idx (b, 4)]) * third;
  qim[idx (b, 2)] = (4.0*qim[idx (b, 3)]-qim[idx (b, 4)]) * third;
  qre[idx (b, N - 3)] = (4.0*qre[idx (b, N-4)]-qre[idx (b, N-5)]) * third;
  qim[idx (b, N - 3)] = (4.0*qim[idx (b, N-4)]-qim[idx (b, N-5)]) * third;

  qre[idx (b, 1)] = (4.0*qre[idx (b, 2)]-qre[idx (b, 3)]) * third;
  qim[idx (b, 1)] = (4.0*qim[idx (b, 2)]-qim[idx (b, 3)]) * third;
  qre[idx (b, N - 2)] = (4.0*qre[idx (b, N-3)]-qre[idx (b, N-4)]) * third;
  qim[idx (b, N - 2)] = (4.0*qim[idx (b, N-3)]-qim[idx (b, N-4)]) * third;

  qre[idx (b, 0)] = (4.0*qre[idx (b, 1)]-qre[idx (b, 2)]) * third;
  qim[idx (b, 0)] = (4.0*qim[idx (b, 1)]-qim[idx (b, 2)]) * third;
  qre[idx (b, N - 1)] = (4.0*qre[idx (b, N-2)]-qre[idx (b, N-3)]) * third;
  qim[idx (b, N - 1)] = (4.0*qim[idx (b, N-2)]-qim[idx (b, N-3)]) * third;

  pre[idx (b, 2)] = (4.0*pre[idx (b, 3)]-pre[idx (b, 4)]) * third;
  pim[idx (b, 2)] = (4.0*pim[idx (b, 3)]-pim[idx (b, 4)]) * third;
  pre[idx (b, N - 3)] = (4.0*pre[idx (b, N-4)]-pre[idx (b, N-5)]) * third;
  pim[idx (b, N - 3)] = (4.0*pim[idx (b, N-4)]-pim[idx (b, N-5)]) * third;

  pre[idx (b, 1)] = (4.0*pre[idx (b, 2)]-pre[idx (b, 3)]) * third;
  pim[idx (b, 1)] = (4.0*pim[idx (b, 2)]-pim[idx (b, 3)]) * third;
  pre[idx (b, N - 2)] = (4.0*pre[idx (b, N-3)]-pre[idx (b, N-4)]) * third;
  pim[idx (b, N - 2)] = (4.0*pim[idx (b, N-3)]-pim[idx (b, N-4)]) * third;

  pre[idx (b, 0)] = (4.0*pre[idx (b, 1)]-pre[idx (b, 2)]) * third;
  pim[idx (b, 0)] = (4.0*pim[idx (b, 1)]-pim[idx (b, 2)]) * third;
  pre[idx (b, N - 1)] = (4.0*pre[idx (b, N-2)]-pre[idx (b, N-3)]) * third;
  pim[idx (b, N - 1)] = (4.0*pim[idx (b, N-2)]-pim[idx (b, N-3)]) * third;

  d_qre_buff[b] = qre[idx (b, N - 1)];
  d_qim_buff[b] = qim[idx (b, N - 1)];

}

__global__ void
reset_sourced ()

{
  int b;
  int a;

  b = blockIdx.y * blockDim.y + threadIdx.y;
  a = blockIdx.x * blockDim.x + threadIdx.x;

  tred[b][a] = 0.0;
  timd[b][a] = 0.0;

}

__global__ void
sourced (double *theta, double start, double timer, double rp, double phip, double tp,
	 double E, double lz, double Q,
	 double drdt, double d2rdt2, double d3rdt3, double dthdt, 
	 double d2thdt2, double d3thdt3, double dphidt, double d2phidt2, double *rr)
{

  cudacomplex i;
  int j, k, ip, it;
  double r, th, wt, wr, stheta, ctheta, pie;
  double xp, cs2, delta, wp;
  double a, nmu, rpl, rm;
  double DelR, DelR1, DelR2;
  double DelTH, DelTH1, DelTH2;

  cudacomplex t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12, t13, t14,
    t15, t16, t17, t18, t19, t20, t21, t22, t23, t24, t25, t26, t27, t28, t29,
    t30, t31, t32, t33, t34, t35, t36, t37, t38, t39, t40, t41, t42, t43, t44,
    t45, t46, t47, t48, t49, t50, t51, t52, t53, t54, t55, t56, t57, t58, t59,
    t60, t61, t62, t63, t64, t65, t66, t67, t68, t69, t70, t71, t72, t73, t74,
    t75, t76, t77, t78, t79, t80, t81, t82, t83, t84, t85, t86, t87, t88, t89,
    t90, t91, t92, t93, t94, t95, t96, t97, t98, t99, t100, t101, t102, t103,
    t104, t105, t106, t107, t108, t109, t110, t111, t112, t113, t114, t115,
    t116, t117, t118, t119, t120, t121, t122, t123, t124, t125, t126, t127,
    t128, t129, t130, t131, t132, t133, t134, t135, t136, t137, t138, t139,
    t140, t141, t142, t143, t144, t145, t146, t147, t148, t149, t150, t151,
    t152, t153, t154, t155, t156, t157, t158, t159, t160, t161, t162, t163,
    t164, t165, t166, t167, t168, t169, t170, t171, t172, t173, t174, t175,
    t176, t177, t178, t179, t180, t181, t182, t183, t184, t185, t186, t187,
    t188, t189, t190, t191, t192, t193, t194, t195, t196, t197, t198, t199,
    t200, t201, t202, t203, t204, t205, t206, t207, t208, t209, t210, t211,
    t212, t213, t214, t215, t216, t217, t218, t219, t220, t221, t222, t223,
    t224, t225, t226, t227, t228, t229, t230, t231, t232, t233, t234, t235,
    t236, t237, t238, t239, t240, t241, t242, t243, t244, t245, t246, t247,
    t248, t249, t250, t251, t252, t253, t254, t255, t256, t257, t258, t259,
    t260, t261, t262, t263, t264, t265, t266, t267, t268, t269, t270, t271,
    t272, t273, t274, t275, t276, t277, t278, t279, t280, t281, t282, t283,
    t284, t285, t286, t287, t288, t289, t290, t291, t292, t293, t294, t295,
    t296, t297, t298, t299, t300, t301, t302, t303, t304, t305, t306, t307,
    t308, t309, t310, t311, t312, t313, t314, t315, t316, t317, t318, t319,
    t320, t321, t322, t323, t324, t325, t326, t327, t328, t329, t330, t331,
    t332, t333, t334, t335, t336, t337, t338, t339, t340, t341, t342, t343,
    t344, t345, t346, t347, t348, t349, t350, t351, t352, t353, t354, t355,
    t356, t357, t358, t359, t360, t361, t362, t363, t364, t365, t366, t367,
    t368, t369, t370, t371, t372, t373, t374, t375, t376, t377, t378, t379,
    t380, t381, t382, t383, t384, t385, t386, t387, t388, t389, t390, t391,
    t392, t393, t394, t395, t396, t397, t398, t399, t400, t401, t402, t403,
    t404, t405, t406, t407, t408, t409, t410, t411, t412, t413, t414, t415,
    t416, t417, t418, t419, t420, t421, t422, t423, t424, t425, t426, t427,
    t428, t429, t430, t431, t432, t433, t434, t435, t436, t437, t438, t439,
    t440, t441, t442, t443, t444, t445, t446, t447, t448, t449, t450, t451,
    t452, t453, t454, t455, t456, t457, t458, t459, t460, t461, t462, t463,
    t464, t465, t466, t467, t468, t469, t470, t471, t472, t473, t474, t475,
    t476, t477, t478, t479, t480, t481, t482, t483, t484, t485, t486, t487,
    t488, t489, t490, t491, t492, t493, t494, t495, t496, t497, t498, t499,
    t500, t501, t502, t503, t504, t505, t506, t507, t508, t509, t510, t511,
    t512, t513, t514, t515, t516, t517, t518, t519, t520, t521, t522, t523,
    t524, t525, t526, t527, t528, t529, t530, t531, t532, t533, t534, t535,
    t536, t537, t538, t539, t540, t541, t542, t543, t544, t545, t546, t547,
    t548, t549, t550, t551, t552, t553, t554, t555, t556, t557, t558, t559,
    t560, t561, t562, t563, t564, t565, t566, t567, t568, t569, t570, t571,
    t572, t573, t574, t575, t576, t577, t578, t579, t580, t581, t582, t583,
    t584, t585, t586, t587, t588, t589, t590, t591, t592, t593, t594, t595,
    t596, t597, t598, t599, t600, t601, t602, t603, t604, t605, t606, t607,
    t608, t609, t610, t611, t612, t613, t614, t615, t616, t617, t618, t619,
    t620, t621, t622, t623, t624, t625, t626, t627, t628, t629, t630, t631,
    t632, t633, t634, t635, t636, t637, t638, t639, t640, t641, t642, t643,
    t644, t645, t646, t647, t648, t649, t650, t651, t652, t653, t654, t655,
    t656, t657, t658, t659, t660, t661, t662, t663, t664, t665, t666, t667,
    t668, t669, t670, t671, t672, t673, t674, t675, t676, t677, t678, t679,
    t680, t681, t682, t683, t684, t685, t686, t687, t688, t689, t690, t691,
    t692, t693, t694, t695, t696, t697, t698, t699, t700, t701, t702, t703,
    t704, t705, t706, t707, t708, t709, t710, t711, t712, t713, t714, t715,
    t716, t717, t718, t719, t720, t721, t722, t723, t724, t725, t726, t727,
    t728, t729, t730, t731, t732, t733, t734, t735, t736, t737, t738, t739,
    t740, t741, t742, t743, t744, t745, t746, t747, t748, t749, t750, t751,
    t752, t753, t754, t755, t756, t757, t758, t759, t760, t761, t762, t763,
    t764, t765, t766, t767, t768, t769, t770, t771, t772, t773, t774, t775,
    t776, t777, t778, t779, t780, t781, t782, t783, t784, t785, t786, t787,
    t788, t789, t790, t791, t792, t793, t794, t795, t796, t797, t798, t799,
    t800, t801, t802, t803, t804, t805, t806, t807, t808, t809, t810, t811,
    t812, t813, t814, t815, t816, t817, t818, t819, t820, t821, t822, t823,
    t824, t825, t826, t827, t828, t829, t830, t831, t832, t833, t834, t835,
    t836, t837, t838, t839, t840, t841, t842, t843, t844, t845, t846, t847,
    t848, t849, t850, t851, t852, t853, t854, t855, t856, t857, t858, t859,
    t860, t861, t862, t863, t864, t865, t866, t867, t868, t869, t870, t871,
    t872, t873, t874, t875, t876, t877, t878, t879, t880, t881, t882, t883,
    t884, t885, t886, t887, t888, t889, t890, t891, t892, t893, t894, t895,
    t896, t897, t898, t899, t900, t901, t902, t903, t904, t905, t906, t907,
    t908, t909, t910, t911, t912, t913, t914, t915, t916, t917, t918, t919,
    t920, t921, t922, t923, t924, t925, t926, t927, t928, t929, t930, t931,
    t932, t933, t934, t935, t936, t937, t938, t939, t940, t941, t942, t943,
    t944, t945, t946, t947, t948, t949, t950, t951, t952, t953, t954, t955,
    t956, t957, t958, t959, t960, t961, t962, t963, t964, t965, t966, t967,
    t968, t969, t970, t971, t972, t973, t974, t975, t976, t977, t978, t979,
    t980, t981, t982, t983, t984, t985, t986, t987, t988, t989, t990, t991,
    t992, t993, t994, t995, t996, t997, t998, t999, t1000, t1001, t1002,
    t1003, t1004, t1005, t1006, t1007, t1008, t1009, t1010, t1011, t1012,
    t1013, t1014, t1015, t1016, t1017, t1018, t1019, t1020, t1021, t1022,
    t1023, t1024, t1025, t1026, t1027, t1028, t1029, t1030, t1031, t1032,
    t1033, t1034, t1035, t1036, t1037, t1038, t1039, t1040, t1041, t1042,
    t1043, t1044, t1045, t1046, t1047, t1048, t1049, t1050, t1051, t1052,
    t1053, t1054, t1055, t1056, t1057, t1058, t1059, t1060, t1061, t1062,
    t1063, t1064, t1065, t1066, t1067, t1068, t1069, t1070, t1071, t1072,
    t1073, t1074, t1075, t1076, t1077, t1078, t1079, t1080, t1081, t1082,
    t1083, t1084, t1085, t1086, t1087, t1088, t1089, t1090, t1091, t1092,
    t1093, t1094, t1095, t1096, t1097, t1098, t1099, t1100, t1101, t1102,
    t1103, t1104, t1105, t1106, t1107, t1108, t1109, t1110, t1111, t1112,
    t1113, t1114, t1115, t1116, t1117, t1118, t1119, t1120, t1121, t1122,
    t1123, t1124, t1125, t1126, t1127, t1128, t1129, t1130, t1131, t1132,
    t1133, t1134, t1135, t1136, t1137, t1138, t1139, t1140, t1141, t1142,
    t1143, t1144, t1145, t1146, t1147, t1148, t1149, t1150, t1151, t1152,
    t1153, t1154, t1155, t1156, t1157, t1158, t1159, t1160, t1161, t1162,
    t1163, t1164, t1165, t1166, t1167, t1168, t1169, t1170, t1171, t1172,
    t1173, t1174, t1175, t1176, t1177, t1178, t1179, t1180, t1181, t1182,
    t1183, t1184, t1185, t1186, t1187, t1188, t1189, t1190, t1191, t1192,
    t1193, t1194, t1195, t1196, t1197, t1198, t1199, t1200, t1201, t1202,
    t1203, t1204, t1205, t1206, t1207, t1208, t1209, t1210, t1211, t1212,
    t1213, t1214, t1215, t1216, t1217, t1218, t1219, t1220, t1221, t1222,
    t1223, t1224, t1225, t1226, t1227, t1228, t1229, t1230, t1231, t1232,
    t1233, t1234, t1235, t1236, t1237, t1238, t1239, t1240, t1241, t1242,
    t1243, t1244, t1245, t1246, t1247, t1248, t1249, t1250, t1251, t1252,
    t1253, t1254, t1255, t1256, t1257, t1258, t1259, t1260, t1261, t1262,
    t1263, t1264, t1265, t1266, t1267, t1268, t1269, t1270, t1271, t1272,
    t1273, t1274, t1275, t1276, t1277, t1278, t1279, t1280, t1281, t1282,
    t1283, t1284, t1285, t1286, t1287, t1288, t1289, t1290, t1291, t1292,
    t1293, t1294, t1295, t1296, t1297, t1298, t1299, t1300, t1301, t1302,
    t1303, t1304, t1305, t1306, t1307, t1308, t1309, t1310, t1311, t1312,
    t1313, t1314, t1315, t1316, t1317, t1318, t1319, t1320, t1321, t1322,
    t1323, t1324, t1325, t1326, t1327, t1328, t1329, t1330, t1331, t1332,
    t1333, t1334, t1335, t1336, t1337, t1338, t1339, t1340, t1341, t1342,
    t1343, t1344, t1345, t1346, t1347, t1348, t1349, t1350, t1351, t1352,
    t1353, t1354, t1355, t1356, t1357, t1358, t1359, t1360, t1361, t1362,
    t1363, t1364, t1365, t1366, t1367, t1368, t1369, t1370, t1371, t1372,
    t1373, t1374, t1375, t1376, t1377, t1378, t1379, t1380, t1381, t1382,
    t1383, t1384, t1385, t1386, t1387, t1388, t1389, t1390, t1391, t1392,
    t1393, t1394, t1395, t1396, t1397, t1398, t1399, t1400, t1401, t1402,
    t1403, t1404, t1405, t1406, t1407, t1408, t1409, t1410, t1411, t1412,
    t1413, t1414, t1415, t1416, t1417, t1418, t1419, t1420, t1421, t1422,
    t1423, t1424, t1425, t1426, t1427, t1428, t1429, t1430, t1431, t1432,
    t1433, t1434, t1435, t1436, t1437, t1438, t1439, t1440, t1441, t1442,
    t1443, t1444, t1445, t1446, t1447, t1448, t1449, t1450, t1451, t1452,
    t1453, t1454, t1455, t1456, t1457, t1458, t1459, t1460, t1461, t1462,
    t1463, t1464, t1465, t1466, t1467, t1468, t1469, t1470, t1471, t1472,
    t1473, t1474, t1475, t1476, t1477, t1478, t1479, t1480, t1481, t1482,
    t1483, t1484, t1485, t1486, t1487, t1488, t1489, t1490, t1491, t1492,
    t1493, t1494, t1495, t1496, t1497, t1498, t1499, t1500, t1501, t1502,
    t1503, t1504, t1505, t1506, t1507, t1508, t1509, t1510, t1511, t1512,
    t1513, t1514, t1515, t1516, t1517, t1518, t1519, t1520, t1521, t1522,
    t1523, t1524, t1525, t1526, t1527, t1528, t1529, t1530, t1531, t1532,
    t1533, t1534, t1535, t1536, t1537, t1538, t1539, t1540, t1541, t1542,
    t1543, t1544, t1545, t1546, t1547, t1548, t1549, t1550, t1551, t1552,
    t1553, t1554, t1555, t1556, t1557, t1558, t1559, t1560, t1561, t1562,
    t1563, t1564, t1565, t1566, t1567, t1568, t1569, t1570, t1571, t1572,
    t1573, t1574, t1575, t1576, t1577, t1578, t1579, t1580, t1581, t1582,
    t1583, t1584, t1585, t1586, t1587, t1588, t1589, t1590, t1591, t1592,
    t1593, t1594, t1595, t1596, t1597, t1598, t1599, t1600, t1601, t1602,
    t1603, t1604, t1605, t1606, t1607, t1608, t1609, t1610, t1611, t1612,
    t1613, t1614, t1615, t1616, t1617, t1618, t1619, t1620, t1621, t1622,
    t1623, t1624, t1625, t1626, t1627, t1628, t1629, t1630, t1631, t1632,
    t1633, t1634, t1635, t1636, t1637, t1638, t1639, t1640, t1641, t1642,
    t1643, t1644, t1645, t1646, t1647, t1648, t1649, t1650, t1651, t1652,
    t1653, t1654, t1655, t1656, t1657, t1658, t1659, t1660, t1661, t1662,
    t1663, t1664, t1665, t1666, t1667, t1668, t1669, t1670, t1671, t1672,
    t1673, t1674, t1675, t1676, t1677, t1678, t1679, t1680, t1681, t1682,
    t1683, t1684, t1685, t1686, t1687, t1688, t1689, t1690, t1691, t1692,
    t1693, t1694, t1695, t1696, t1697, t1698, t1699, t1700, t1701, t1702,
    t1703, t1704, t1705, t1706, t1707, t1708, t1709, t1710, t1711, t1712,
    t1713, t1714, t1715, t1716, t1717, t1718, t1719, t1720, t1721, t1722,
    t1723, t1724, t1725, t1726, t1727, t1728, t1729, t1730, t1731, t1732,
    t1733, t1734, t1735, t1736, t1737, t1738, t1739, t1740, t1741, t1742,
    t1743, t1744, t1745, t1746, t1747, t1748, t1749, t1750, t1751, t1752,
    t1753, t1754, t1755, t1756, t1757, t1758, t1759, t1760, t1761, t1762,
    t1763, t1764, t1765, t1766, t1767, t1768, t1769, t1770, t1771, t1772,
    t1773, t1774, t1775, t1776, t1777, t1778, t1779, t1780, t1781, t1782,
    t1783, t1784, t1785, t1786, t1787, t1788, t1789, t1790, t1791, t1792,
    t1793, t1794, t1795, t1796, t1797, t1798, t1799, t1800, t1801, t1802,
    t1803, t1804, t1805, t1806, t1807, t1808, t1809, t1810, t1811, t1812,
    t1813, t1814, t1815, t1816, t1817, t1818, t1819, t1820, t1821, t1822,
    t1823, t1824, t1825, t1826, t1827, t1828, t1829, t1830, t1831, t1832,
    t1833, t1834, t1835, t1836, t1837, t1838, t1839, t1840, t1841, t1842,
    t1843, t1844, t1845, t1846, t1847, t1848, t1849, t1850, t1851, t1852,
    t1853, t1854, t1855, t1856, t1857, t1858, t1859, t1860, t1861, t1862,
    t1863, t1864, t1865, t1866, t1867, t1868, t1869, t1870, t1871, t1872,
    t1873, t1874, t1875, t1876, t1877, t1878, t1879, t1880, t1881, t1882,
    t1883, t1884, t1885, t1886, t1887, t1888, t1889, t1890, t1891, t1892,
    t1893, t1894, t1895, t1896, t1897, t1898, t1899, t1900, t1901, t1902,
    t1903, t1904, t1905, t1906, t1907, t1908, t1909, t1910, t1911, t1912,
    t1913, t1914, t1915, t1916, t1917, t1918, t1919, t1920, t1921, t1922,
    t1923, t1924, t1925, t1926, t1927, t1928, t1929, t1930, t1931, t1932,
    t1933, t1934, t1935, t1936, t1937, t1938, t1939, t1940, t1941, t1942,
    t1943, t1944, t1945, t1946, t1947, t1948, t1949, t1950, t1951, t1952,
    t1953, t1954, t1955, t1956, t1957, t1958, t1959, t1960, t1961, t1962,
    t1963, t1964, t1965, t1966, t1967, t1968, t1969, t1970, t1971, t1972,
    t1973, t1974, t1975, t1976, t1977, t1978, t1979, t1980, t1981, t1982,
    t1983, t1984, t1985, t1986, t1987, t1988, t1989, t1990, t1991, t1992,
    t1993, t1994, t1995, t1996, t1997, t1998, t1999, t2000, t2001, t2002,
    t2003, t2004, t2005, t2006, t2007, t2008, t2009, t2010, t2011, t2012,
    t2013, t2014, t2015, t2016, t2017, t2018, t2019, t2020, t2021, t2022,
    t2023, t2024, t2025, t2026, t2027, t2028, t2029, t2030, t2031, t2032,
    t2033, t2034, t2035, t2036, t2037, t2038, t2039, t2040, t2041, t2042,
    t2043, t2044, t2045, t2046, t2047, t2048, t2049, t2050, t2051, t2052,
    t2053, t2054, t2055, t2056, t2057, t2058, t2059, t2060, t2061, t2062,
    t2063, t2064, t2065, t2066, t2067, t2068, t2069, t2070, t2071, t2072,
    t2073, t2074, t2075, t2076, t2077, t2078, t2079, t2080, t2081, t2082,
    t2083, t2084, t2085, t2086, t2087, t2088, t2089, t2090, t2091, t2092,
    t2093, t2094, t2095, t2096, t2097, t2098, t2099, t2100, t2101, t2102,
    t2103, t2104, t2105, t2106, t2107, t2108, t2109, t2110, t2111, t2112,
    t2113, t2114, t2115, t2116, t2117, t2118, t2119, t2120, t2121, t2122,
    t2123, t2124, t2125, t2126, t2127, t2128, t2129, t2130, t2131, t2132,
    t2133, t2134, t2135, t2136, t2137, t2138, t2139, t2140, t2141, t2142,
    t2143, t2144, t2145, t2146, t2147, t2148, t2149, t2150, t2151, t2152,
    t2153, t2154, t2155, t2156, t2157, t2158, t2159, t2160, t2161, t2162,
    t2163, t2164, t2165, t2166, t2167, t2168, t2169, t2170, t2171, t2172,
    t2173, t2174, t2175, t2176, t2177, t2178, t2179, t2180, t2181, t2182,
    t2183, t2184, t2185, t2186, t2187, t2188, t2189, t2190, t2191, t2192,
    t2193, t2194, t2195, t2196, t2197, t2198, t2199, t2200, t2201, t2202,
    t2203, t2204, t2205, t2206, t2207, t2208, t2209, t2210, t2211, t2212,
    t2213, t2214, t2215, t2216, t2217, t2218, t2219, t2220, t2221, t2222,
    t2223, t2224, t2225, t2226, t2227, t2228, t2229, t2230, t2231, t2232,
    t2233, t2234, t2235, t2236, t2237, t2238, t2239, t2240, t2241, t2242,
    t2243, t2244, t2245, t2246, t2247, t2248, t2249, t2250, t2251, t2252,
    t2253, t2254, t2255, t2256, t2257, t2258, t2259, t2260, t2261, t2262,
    t2263, t2264, t2265, t2266, t2267, t2268, t2269, t2270, t2271, t2272,
    t2273, t2274, t2275, t2276, t2277, t2278, t2279, t2280, t2281, t2282,
    t2283, t2284, t2285, t2286, t2287, t2288, t2289, t2290, t2291, t2292,
    t2293, t2294, t2295, t2296, t2297, t2298, t2299, t2300, t2301, t2302,
    t2303, t2304, t2305, t2306, t2307, t2308, t2309, t2310, t2311, t2312,
    t2313, t2314, t2315, t2316, t2317, t2318, t2319, t2320, t2321, t2322,
    t2323, t2324, t2325, t2326, t2327, t2328, t2329, t2330, t2331, t2332,
    t2333, t2334, t2335, t2336, t2337, t2338, t2339, t2340, t2341, t2342,
    t2343, t2344, t2345, t2346, t2347, t2348, t2349, t2350, t2351, t2352,
    t2353, t2354, t2355, t2356, t2357, t2358, t2359, t2360, t2361, t2362,
    t2363, t2364, t2365, t2366, t2367, t2368, t2369, t2370, t2371, t2372,
    t2373, t2374, t2375, t2376, t2377, t2378, t2379, t2380, t2381, t2382,
    t2383, t2384, t2385, t2386, t2387, t2388, t2389, t2390, t2391, t2392,
    t2393, t2394, t2395, t2396, t2397, t2398, t2399, t2400, t2401, t2402,
    t2403, t2404, t2405, t2406, t2407, t2408, t2409, t2410, t2411, t2412,
    t2413, t2414, t2415, t2416, t2417, t2418, t2419, t2420, t2421, t2422,
    t2423, t2424, t2425, t2426, t2427, t2428, t2429, t2430, t2431, t2432,
    t2433, t2434, t2435, t2436, t2437, t2438, t2439, t2440, t2441, t2442,
    t2443, t2444, t2445, t2446, t2447, t2448, t2449, t2450, t2451, t2452,
    t2453, t2454, t2455, t2456, t2457, t2458, t2459, t2460, t2461, t2462,
    t2463, t2464, t2465, t2466, t2467, t2468, t2469, t2470, t2471, t2472,
    t2473, t2474, t2475, t2476, t2477, t2478, t2479, t2480, t2481, t2482,
    t2483, t2484, t2485, t2486, t2487, t2488, t2489, t2490, t2491, t2492,
    t2493, t2494, t2495, t2496, t2497, t2498, t2499, t2500, t2501, t2502,
    t2503, t2504, t2505, t2506, t2507, t2508, t2509, t2510, t2511, t2512,
    t2513, t2514, t2515, t2516, t2517, t2518, t2519, t2520, t2521, t2522,
    t2523, t2524, t2525, t2526, t2527, t2528, t2529, t2530, t2531, t2532,
    t2533, t2534, t2535, t2536, t2537, t2538, t2539, t2540, t2541, t2542,
    t2543, t2544, t2545, t2546, t2547, t2548, t2549, t2550, t2551, t2552,
    t2553, t2554, t2555, t2556, t2557, t2558, t2559, t2560, t2561, t2562,
    t2563, t2564, t2565, t2566, t2567, t2568, t2569, t2570, t2571, t2572,
    t2573, t2574, t2575, t2576, t2577, t2578, t2579, t2580, t2581, t2582,
    t2583, t2584, t2585, t2586, t2587, t2588, t2589, t2590, t2591, t2592,
    t2593, t2594, t2595, t2596, t2597, t2598, t2599, t2600, t2601, t2602,
    t2603, t2604, t2605, t2606, t2607, t2608, t2609, t2610, t2611, t2612,
    t2613, t2614, t2615, t2616, t2617, t2618, t2619, t2620, t2621, t2622,
    t2623, t2624, t2625, t2626, t2627, t2628, t2629, t2630, t2631, t2632,
    t2633, t2634, t2635, t2636, t2637, t2638, t2639, t2640, t2641, t2642,
    t2643, t2644, t2645, t2646, t2647, t2648, t2649, t2650, t2651, t2652,
    t2653, t2654, t2655, t2656, t2657, t2658, t2659, t2660, t2661, t2662,
    t2663, t2664, t2665, t2666, t2667, t2668, t2669, t2670, t2671, t2672,
    t2673, t2674, t2675, t2676, t2677, t2678, t2679, t2680, t2681, t2682,
    t2683, t2684, t2685, t2686, t2687, t2688, t2689, t2690, t2691, t2692,
    t2693, t2694, t2695, t2696, t2697, t2698, t2699, t2700, t2701, t2702,
    t2703, t2704, t2705, t2706, t2707, t2708, t2709, t2710, t2711, t2712,
    t2713, t2714, t2715, t2716, t2717, t2718, t2719, t2720, t2721, t2722,
    t2723, t2724, t2725, t2726, t2727, t2728, t2729, t2730, t2731, t2732,
    t2733, t2734, t2735, t2736, t2737, t2738, t2739, t2740, t2741, t2742,
    t2743, t2744, t2745, t2746, t2747, t2748, t2749, t2750, t2751, t2752,
    t2753, t2754, t2755, t2756, t2757, t2758, t2759, t2760, t2761, t2762,
    t2763, t2764, t2765, t2766, t2767, t2768, t2769, t2770, t2771, t2772,
    t2773, t2774, t2775, t2776, t2777, t2778, t2779, t2780, t2781, t2782,
    t2783, t2784, t2785, t2786, t2787, t2788, t2789, t2790, t2791, t2792,
    t2793, t2794, t2795, t2796, t2797, t2798, t2799, t2800, t2801, t2802,
    t2803, t2804, t2805, t2806, t2807, t2808, t2809, t2810, t2811, t2812,
    t2813, t2814, t2815, t2816, t2817, t2818, t2819, t2820, t2821, t2822,
    t2823, t2824, t2825, t2826, t2827, t2828, t2829, t2830, t2831, t2832,
    t2833, t2834, t2835, t2836, t2837, t2838, t2839, t2840, t2841, t2842,
    t2843, t2844, t2845, t2846, t2847, t2848, t2849, t2850, t2851, t2852,
    t2853, t2854, t2855, t2856, t2857, t2858, t2859, t2860, t2861, t2862,
    t2863, t2864, t2865, t2866, t2867, t2868, t2869, t2870, t2871, t2872,
    t2873, t2874, t2875, t2876, t2877, t2878, t2879, t2880, t2881, t2882,
    t2883, t2884, t2885, t2886, t2887, t2888, t2889, t2890, t2891, t2892,
    t2893, t2894, t2895, t2896, t2897, t2898, t2899, t2900, t2901, t2902,
    t2903, t2904, t2905, t2906, t2907, t2908, t2909, t2910, t2911, t2912,
    t2913, t2914, t2915, t2916, t2917, t2918, t2919, t2920, t2921, t2922,
    t2923, t2924, t2925, t2926, t2927, t2928, t2929, t2930, t2931, t2932,
    t2933, t2934, t2935, t2936, t2937, t2938, t2939, t2940, t2941, t2942,
    t2943, t2944, t2945, t2946, t2947, t2948, t2949, t2950, t2951, t2952,
    t2953, t2954, t2955, t2956, t2957, t2958, t2959, t2960, t2961, t2962,
    t2963, t2964, t2965, t2966, t2967, t2968, t2969, t2970, t2971, t2972,
    t2973, t2974, t2975, t2976, t2977, t2978, t2979, t2980, t2981, t2982,
    t2983, t2984, t2985, t2986, t2987, t2988, t2989, t2990, t2991, t2992,
    t2993, t2994, t2995, t2996, t2997, t2998, t2999, t3000, t3001, t3002,
    t3003, t3004, t3005, t3006, t3007, t3008, t3009, t3010, t3011, t3012,
    t3013, t3014, t3015, t3016, t3017, t3018, t3019, t3020, t3021, t3022,
    t3023, t3024, t3025, t3026, t3027, t3028, t3029, t3030, t3031, t3032,
    t3033, t3034, t3035, t3036, t3037, t3038, t3039, t3040, t3041, t3042,
    t3043, t3044, t3045, t3046, t3047, t3048, t3049, t3050, t3051, t3052,
    t3053, t3054, t3055, t3056, t3057, t3058, t3059, t3060, t3061, t3062,
    t3063, t3064, t3065, t3066, t3067, t3068, t3069, t3070, t3071, t3072,
    t3073, t3074, t3075, t3076, t3077, t3078, t3079, t3080, t3081, t3082,
    t3083, t3084, t3085, t3086, t3087, t3088, t3089, t3090, t3091, t3092,
    t3093, t3094, t3095, t3096, t3097, t3098, t3099, t3100, t3101, t3102,
    t3103, t3104, t3105, t3106, t3107, t3108, t3109, t3110, t3111, t3112,
    t3113, t3114, t3115, t3116, t3117, t3118, t3119, t3120, t3121, t3122,
    t3123, t3124, t3125, t3126, t3127, t3128, t3129, t3130, t3131, t3132,
    t3133, t3134, t3135, t3136, t3137, t3138, t3139, t3140, t3141, t3142,
    t3143, t3144, t3145, t3146, t3147, t3148, t3149, t3150, t3151, t3152,
    t3153, t3154, t3155, t3156, t3157, t3158, t3159, t3160, t3161, t3162,
    t3163, t3164, t3165, t3166, t3167, t3168, t3169, t3170, t3171, t3172,
    t3173, t3174, t3175, t3176, t3177, t3178, t3179, t3180, t3181, t3182,
    t3183, t3184, t3185, t3186, t3187, t3188, t3189, t3190, t3191, t3192,
    t3193, t3194, t3195, t3196, t3197, t3198, t3199, t3200, t3201, t3202,
    t3203, t3204, t3205, t3206, t3207, t3208, t3209, t3210, t3211, t3212,
    t3213, t3214, t3215, t3216, t3217, t3218, t3219, t3220, t3221, t3222,
    t3223, t3224, t3225, t3226, t3227, t3228, t3229, t3230, t3231, t3232,
    t3233, t3234, t3235, t3236, t3237, t3238, t3239, t3240, t3241, t3242,
    t3243, t3244, t3245, t3246, t3247, t3248, t3249, t3250, t3251, t3252,
    t3253, t3254, t3255, t3256, t3257, t3258, t3259, t3260, t3261, t3262,
    t3263, t3264, t3265, t3266, t3267, t3268, t3269, t3270, t3271, t3272,
    t3273, t3274, t3275, t3276, t3277, t3278, t3279, t3280, t3281, t3282,
    t3283, t3284, t3285, t3286, t3287, t3288, t3289, t3290, t3291, t3292,
    t3293, t3294, t3295, t3296, t3297, t3298, t3299, t3300, t3301, t3302,
    t3303, t3304, t3305, t3306, t3307, t3308, t3309, t3310, t3311, t3312,
    t3313, t3314, t3315, t3316, t3317, t3318, t3319, t3320, t3321, t3322,
    t3323, t3324, t3325, t3326, t3327, t3328, t3329, t3330, t3331, t3332,
    t3333, t3334, t3335, t3336, t3337, t3338, t3339, t3340, t3341, t3342,
    t3343, t3344, t3345, t3346, t3347, t3348, t3349, t3350, t3351, t3352,
    t3353, t3354, t3355, t3356, t3357, t3358, t3359, t3360, t3361, t3362,
    t3363, t3364, t3365, t3366, t3367, t3368, t3369, t3370, t3371, t3372,
    t3373, t3374, t3375, t3376, t3377, t3378, t3379, t3380, t3381, t3382,
    t3383, t3384, t3385, t3386, t3387, t3388, t3389, t3390, t3391, t3392,
    t3393, t3394, t3395, t3396, t3397, t3398, t3399, t3400, t3401, t3402,
    t3403, t3404, t3405, t3406, t3407, t3408, t3409, t3410, t3411, t3412,
    t3413, t3414, t3415, t3416, t3417, t3418, t3419, t3420, t3421, t3422,
    t3423, t3424, t3425, t3426, t3427, t3428, t3429, t3430, t3431, t3432,
    t3433, t3434, t3435, t3436, t3437, t3438, t3439, t3440, t3441, t3442,
    t3443, t3444, t3445, t3446, t3447, t3448, t3449, t3450, t3451, t3452,
    t3453, t3454, t3455, t3456, t3457, t3458, t3459, t3460, t3461, t3462,
    t3463, t3464, t3465, t3466, t3467, t3468, t3469, t3470, t3471, t3472,
    t3473, t3474, t3475, t3476, t3477, t3478, t3479, t3480, t3481, t3482,
    t3483, t3484, t3485, t3486, t3487, t3488, t3489, t3490, t3491, t3492,
    t3493, t3494, t3495, t3496, t3497, t3498, t3499, t3500, t3501, t3502,
    t3503, t3504, t3505, t3506, t3507, t3508, t3509, t3510, t3511, t3512,
    t3513, t3514, t3515, t3516, t3517, t3518, t3519, t3520, t3521, t3522,
    t3523, t3524, t3525, t3526, t3527, t3528, t3529, t3530, t3531, t3532,
    t3533, t3534, t3535, t3536, t3537, t3538, t3539, t3540, t3541, t3542,
    t3543, t3544, t3545, t3546, t3547, t3548, t3549, t3550, t3551, t3552,
    t3553, t3554, t3555, t3556, t3557, t3558, t3559, t3560, t3561, t3562,
    t3563, t3564, t3565, t3566, t3567, t3568, t3569, t3570, t3571, t3572,
    t3573, t3574, t3575, t3576, t3577, t3578, t3579, t3580, t3581, t3582,
    t3583, t3584, t3585, t3586, t3587, t3588, t3589, t3590, t3591, t3592,
    t3593, t3594, t3595, t3596, t3597, t3598, t3599, t3600, t3601, t3602,
    t3603, t3604, t3605, t3606, t3607, t3608, t3609, t3610, t3611, t3612,
    t3613, t3614, t3615, t3616, t3617, t3618, t3619, t3620, t3621, t3622,
    t3623, t3624, t3625, t3626, t3627, t3628, t3629, t3630, t3631, t3632,
    t3633, t3634, t3635, t3636, t3637, t3638, t3639, t3640, t3641, t3642,
    t3643, t3644, t3645, t3646, t3647, t3648, t3649, t3650, t3651, t3652,
    t3653, t3654, t3655, t3656, t3657, t3658, t3659, t3660, t3661, t3662,
    t3663, t3664, t3665, t3666, t3667, t3668, t3669, t3670, t3671, t3672,
    t3673, t3674, t3675, t3676, t3677, t3678, t3679, t3680, t3681, t3682,
    t3683, t3684, t3685, t3686, t3687, t3688, t3689, t3690, t3691, t3692,
    t3693, t3694, t3695, t3696, t3697, t3698, t3699, t3700, t3701, t3702,
    t3703, t3704, t3705, t3706, t3707, t3708, t3709, t3710, t3711, t3712,
    t3713, t3714, t3715, t3716, t3717, t3718, t3719, t3720, t3721, t3722,
    t3723, t3724, t3725, t3726, t3727, t3728, t3729, t3730, t3731, t3732,
    t3733, t3734, t3735, t3736, t3737, t3738, t3739, t3740, t3741, t3742,
    t3743, t3744, t3745, t3746, t3747, t3748, t3749, t3750, t3751, t3752,
    t3753, t3754, t3755, t3756, t3757, t3758, t3759, t3760, t3761, t3762,
    t3763, t3764, t3765, t3766, t3767, t3768, t3769, t3770, t3771, t3772,
    t3773, t3774, t3775, t3776, t3777, t3778, t3779, t3780, t3781, t3782,
    t3783, t3784, t3785, t3786, t3787, t3788, t3789, t3790, t3791, t3792,
    t3793, t3794, t3795, t3796, t3797, t3798, t3799, t3800, t3801, t3802,
    t3803, t3804, t3805, t3806, t3807, t3808, t3809, t3810, t3811, t3812,
    t3813, t3814, t3815, t3816, t3817, t3818, t3819, t3820, t3821, t3822,
    t3823, t3824, t3825, t3826, t3827, t3828, t3829, t3830, t3831, t3832,
    t3833, t3834, t3835, t3836, t3837, t3838, t3839, t3840, t3841, t3842,
    t3843, t3844, t3845, t3846, t3847, t3848, t3849, t3850, t3851, t3852,
    t3853, t3854, t3855, t3856, t3857, t3858, t3859, t3860, t3861, t3862,
    t3863, t3864, t3865, t3866, t3867, t3868, t3869, t3870, t3871, t3872,
    t3873, t3874, t3875, t3876, t3877, t3878, t3879, t3880, t3881, t3882,
    t3883, t3884, t3885, t3886, t3887, t3888, t3889, t3890, t3891, t3892,
    t3893, t3894, t3895, t3896, t3897, t3898, t3899, t3900, t3901, t3902,
    t3903, t3904, t3905, t3906, t3907, t3908, t3909, t3910, t3911, t3912,
    t3913, t3914, t3915, t3916, t3917, t3918, t3919, t3920, t3921, t3922,
    t3923, t3924, t3925, t3926, t3927, t3928, t3929, t3930, t3931, t3932,
    t3933, t3934, t3935, t3936, t3937, t3938, t3939, t3940, t3941, t3942,
    t3943, t3944, t3945, t3946, t3947, t3948, t3949, t3950, t3951, t3952,
    t3953, t3954, t3955, t3956, t3957, t3958, t3959, t3960, t3961, t3962,
    t3963, t3964, t3965, t3966, t3967, t3968, t3969, t3970, t3971, t3972,
    t3973, t3974, t3975, t3976, t3977, t3978, t3979, t3980, t3981, t3982,
    t3983, t3984, t3985, t3986, t3987, t3988, t3989, t3990, t3991, t3992,
    t3993, t3994, t3995, t3996, t3997, t3998, t3999;

  cudacomplex mgv1;
  cudacomplex mgv2;
  cudacomplex mgv3;
  cudacomplex mgv4;
  cudacomplex mgv5;
  cudacomplex mgv6;
  cudacomplex mgv7;
  cudacomplex mgv8;
  cudacomplex mgv9;
  cudacomplex mgv10;

  cudacomplex TT;
  cudacomplex rho;

  i.real = 0.0;
  i.img = 1.0;

  j = blockIdx.y * blockDim.y + threadIdx.y;
  k = blockIdx.x * blockDim.x + threadIdx.x;

  pie = 3.141592653589793;
  a = aa;
  nmu = 1.0;
  wt = 6.0 * dtheta;
  wp = 2.0 * dx;

  rpl = mass + sqrt (mass * mass - a * a);
  rm = mass - sqrt (mass * mass - a * a);
  wr = wp * (1.0 - rpl / rp) ;

  xp = rp + (rpl * rpl + aa * aa) / (rpl - rm) * log (rp / rpl - 1.0) -
    (rm * rm + aa * aa) / (rpl - rm) * log (rp / rm - 1.0);

  //xp = rp + 2.0 * mass * log (rp / (2.0 * mass) - 1.0);

  ip = (int) ((xp - Xmin) / dx  );
  it = (int) (tp / dtheta );

  if ( (k + ip - PTSX / 2 > 0) && (j + it - PTSY / 2 > 0) && (j + it - PTSY / 2 < M) ) {

  r = rr[k + ip - PTSX / 2];
  th = theta[j + it - PTSY / 2];

  DelR =
    1.0 / sqrt (2.0 * pie) * exp (-(r - rp) * (r - rp) / (2.0 * wr * wr)) /
    wr;
  DelR1 = -(r - rp) / (wr * wr) * DelR;
  DelR2 =
    (-1.0 / (wr * wr) + (r - rp) * (r - rp) / (wr * wr * wr * wr)) * DelR;


  stheta = sin (th);
  ctheta = cos (th);

  DelTH =
    1.0 / sqrt (2.0 * pie) * exp (-(th - tp) * (th - tp) / (2.0 * wt * wt)) /
    wt;
  DelTH1 = -(th - tp) / (wt * wt) * DelTH;
  DelTH2 =
    (-1.0 / (wt * wt) + (th - tp) * (th - tp) / (wt * wt * wt * wt)) * DelTH;

//    Maple generated expression here 

  t1 = rp * rp;
  t2 = a * a;
  t3 = cos (tp);
  t4 = t3 * t3;
  t5 = t2 * t4;
  t6 = t1 + t5;
  t7 = 1 / t6;
  t8 = t2 + t1;
  t9 = (-1.0 / 4.0 * i) * a;
  t10 = a * t3;
  t11 = rp + (i) * t10;
  t12 = t11 * t11;
  t13 = t9 * t12;
  t14 = t6 * t6;
  t15 = t14 * t14;
  t16 = 1 / t15;
  t17 = (0.0 - i) * a;
  t19 = rp + t17 * t3;
  t20 = t16 * t19;
  t21 = t20 * nmu;
  t22 = t13 * t21;
  t23 = t8 * t8;
  t26 = t1 + t2 - 2.0 * mass * rp;
  t27 = 1 / t26;
  t29 = sin (tp);
  t30 = t29 * t29;
  t31 = t2 * t30;
  t34 = a * lz;
  t35 = t8 * t27;
  t38 = E * (t23 * t27 - t31) + t34 * (1.0 - t35);
  t39 = t38 * t38;
  t40 = 1 / t39;
  t43 = E * t8 + drdt * t38 - t34;
  t44 = t40 * t43;
  t46 = 1 / t30;
  t48 = a * E - lz * t46;
  t51 = (i) * t29 * t48 + dthdt * t38;
  t52 = t44 * t51;
  t53 = 1 / pie;
  t56 = cos (-mm * phip) + i * sin (-mm * phip);
  t57 = t53 * t56;
  t58 = rp * drdt;
  t61 = t26 * t26;
  t62 = 1 / t61;
  t63 = t23 * t62;
  t65 = t58 - mass * drdt;
  t67 = t2 * t3;
  t68 = t29 * dthdt;
  t69 = t67 * t68;
  t75 = t8 * t62;
  t79 =
    E * (4.0 * t35 * t58 - 2.0 * t63 * t65 - 2.0 * t69) +
    t34 * (-2.0 * t58 * t27 + 2.0 * t75 * t65);
  t83 = (1.0 / 2.0 * i) * a;
  t84 = t83 * t12;
  t85 = 1 / t14;
  t86 = t85 * t29;
  t87 = sqrt (2.0);
  t88 = t86 * t87;
  t90 = t11 * t7;
  t91 = 1 / tan (tp);
  t92 = t91 * t87;
  t95 = t84 * t88 - t90 * t92 / 4.0;
  t96 = t95 * nmu;
  t97 = t11 * t85;
  t98 = t97 * t87;
  t99 = t96 * t98;
  t100 = 1 / t38;
  t101 = E * rp;
  t106 = 2.0 * t101 * drdt + d2rdt2 * t38 + drdt * t79;
  t108 = t100 * t106 * t51;
  t109 = 1 / t29;
  t110 = t57 * t109;
  t111 = t108 * t110;
  t114 = (1.0 / 4.0 * i) * a;
  t115 = t12 * t16;
  t116 = t115 * t19;
  t117 = t114 * t116;
  t118 = nmu * t100;
  t120 = t51 * t53;
  t121 = t120 * t56;
  t124 = (-1.0 / 2.0 * i) * a;
  t127 = 1 / t15 / t6;
  t128 = t127 * t19;
  t129 = t128 * nmu;
  t131 = t100 * t43;
  t132 = t131 * t51;
  t133 = t58 - t69;
  t138 = 1 / t14 / t6;
  t139 = t12 * t138;
  t140 = t19 * t26;
  t142 = rp - mass;
  t143 = t19 * t142;
  t144 = t97 * t143;
  t145 = t144 - t139 * t140;
  t146 = t145 * nmu / 2.0;
  t147 = t12 * t85;
  t148 = t147 * t100;
  t149 = t146 * t148;
  t150 = t56 * t109;
  t151 = t3 * dthdt;
  t153 = (2.0 * i) * t46;
  t154 = lz * t3;
  t159 = (i) * t151 * t48 + t153 * t154 * dthdt + d2thdt2 * t38 + dthdt * t79;
  t160 = t150 * t159;
  t161 = t120 * t160;
  t163 = t19 * t19;
  t164 = t163 * t138;
  t165 = t11 * t26;
  t167 = t144 - t164 * t165;
  t168 = t167 * nmu / 2.0;
  t169 = t168 * t148;
  t171 = t97 * t100;
  t173 = t51 * t51;
  t174 = t173 * t53;
  t176 = drdt + t17 * t68;
  t177 = t150 * t176;
  t178 = t174 * t177;
  t180 = t174 * t56;
  t181 = t46 * t3;
  t182 = t181 * dthdt;
  t183 = t180 * t182;
  t186 = t139 * t100;
  t188 = 2.0 * t150 * t133;
  t189 = t174 * t188;
  t191 = t131 * t159;
  t192 = t191 * t110;
  t197 = t163 * t127;
  t198 = t12 * t11;
  t200 = t198 * t26 * nmu;
  t201 = t197 * t200;
  t204 = t150 * t79;
  t208 = t147 * t40;
  t210 = t174 * t204;
  t221 = t12 * t26;
  t222 = t221 * nmu;
  t223 = t197 * t222;
  t224 = t100 * t173;
  t225 = t224 * t53;
  t229 = t96 * t11;
  t230 = t85 * t87;
  t231 = t230 * t100;
  t232 = t229 * t231;
  t233 = t43 * t51;
  t234 = t233 * t53;
  t235 = t56 * t46;
  t236 = t235 * t151;
  t237 = t234 * t236;
  t240 =
    t22 * t52 * t57 * t79 - t99 * t111 / 4.0 +
    t117 * t118 * t106 * t121 +
    2.0 * t124 * t12 * t129 * t132 * t57 * t133 - t149 * t161 +
    t169 * t161 - t146 * t171 * t178 - t169 * t183 / 2.0 +
    t146 * t186 * t189 - t99 * t192 / 4.0 + t149 * t183 / 2.0 -
    t201 * t40 * t173 * t53 * t204 / 8.0 +
    t146 * t208 * t210 / 2.0 - t168 * t208 * t210 / 2.0 +
    t201 * t100 * t51 * t53 * t160 / 4.0 +
    t223 * t225 * t177 / 4.0 + t232 * t237 / 4.0;
  t241 = t230 * t40;
  t243 = t234 * t204;
  t250 = t132 * t57 * t151;
  t251 = t13 * t20 * t109 * nmu * t250;
  t252 = (1.0 / 8.0 * i) * t46;
  t255 = mm * mm;
  t256 = t53 * t255;
  t257 = dphidt * t56;
  t261 = t46 * nmu;
  t263 = t176 * t85 * t87;
  t265 = t53 * mm;
  t266 = t265 * t56;
  t267 = t132 * t266;
  t270 = t261 * t11;
  t271 = t138 * t87;
  t272 = t271 * t100;
  t274 = mm * t56;
  t275 = 2.0 * t274 * t133;
  t280 = t274 * t79;
  t284 = t261 * t98;
  t291 = dthdt * dthdt;
  t294 = t29 * d2thdt2;
  t296 = d2rdt2 + t17 * t3 * t291 + t17 * t294;
  t299 = t233 * t110;
  t302 = nmu * t176;
  t304 = t234 * t188;
  t310 = nmu * t11;
  t311 = t310 * t272;
  t312 = t106 * t51;
  t313 = t312 * t53;
  t317 = t310 * t241;
  t321 = t310 * t85;
  t322 = t87 * t40;
  t323 = t322 * t43;
  t324 = t321 * t323;
  t330 = t310 * t231;
  t331 = t150 * t291;
  t332 = t234 * t331;
  t335 = (1.0 / 4.0 * i) * nmu;
  t336 = t335 * t11;
  t337 = t336 * t231;
  t338 = mm * dphidt;
  t339 = t338 * t150;
  t342 = t16 * t87;
  t345 = 4.0 * t133 * t133;
  t346 = t150 * t345;
  t350 = t310 * t138;
  t352 = 2.0 * t109 * t133;
  t353 = t352 * t79;
  t357 = t87 * t100;
  t358 = t357 * t43;
  t359 = t350 * t358;
  t365 = (-1.0 / 4.0 * i) * nmu;
  t366 = t365 * t11;
  t367 = t230 * t131;
  t368 = t366 * t367;
  t372 = t321 * t358;
  t373 = t120 * t255;
  t374 = dphidt * dphidt;
  t376 = t374 * t56 * t109;
  t381 = 1 / t30 / t29;
  t383 = t381 * t4 * t291;
  t393 = t291 * lz;
  t403 = drdt * drdt;
  t412 = rp * d2rdt2;
  t416 = 1 / t61 / t26;
  t418 = 4.0 * t65 * t65;
  t422 = t403 + t412 - mass * d2rdt2;
  t424 = t31 * t291;
  t426 = t5 * t291;
  t428 = t67 * t294;
  t445 =
    E * (8.0 * t1 * t403 * t27 - 16.0 * t75 * t58 * t65 +
	 4.0 * t35 * t403 + 4.0 * t35 * t412 +
	 2.0 * t23 * t416 * t418 - 2.0 * t63 * t422 + 2.0 * t424 -
	 2.0 * t426 - 2.0 * t428) + t34 * (-2.0 * t403 * t27 -
					   2.0 * t412 * t27 +
					   8.0 * t58 * t62 * t65 -
					   2.0 * t8 * t416 *
					   t418 + 2.0 * t75 * t422);
  t447 =
    (0.0 - i) * t29 * t291 * t48 + (i) * t3 * d2thdt2 * t48 +
    (-2.0 * i) * t4 * t393 * t381 + (-2.0 * i) * t109 * t393 +
    t153 * t154 * d2thdt2 + d3thdt3 * t38 + 2.0 * d2thdt2 * t79 +
    dthdt * t445;
  t452 = t150 * t445;
  t456 =
    -nmu * t296 * t231 * t299 / 8.0 + t302 * t272 * t304 / 2.0 +
    t302 * t241 * t243 / 4.0 + t311 * t313 * t188 / 2.0 +
    t317 * t313 * t204 / 4.0 -
    t324 * t121 * t46 * t79 * t151 / 4.0 - t330 * t332 / 8.0 +
    t337 * t313 * t339 -
    3.0 / 4.0 * t310 * t342 * t100 * t234 * t346 -
    t350 * t323 * t121 * t353 / 2.0 -
    t359 * t121 * t46 * t133 * t151 + t368 * t120 * t338 * t236 +
    t372 * t373 * t376 / 8.0 - t372 * t121 * t383 / 4.0 -
    t330 * t43 * t447 * t110 / 8.0 + t317 * t234 * t452 / 8.0;
  t457 = t403 + t412 + t424 - t426 - t428;
  t458 = 2.0 * t150 * t457;
  t462 = (-1.0 / 2.0 * i) * nmu;
  t466 = t120 * mm;
  t472 = t121 * t182;
  t477 = t234 * t339;
  t479 = t43 * t159;
  t480 = t479 * t53;
  t484 = 1 / t39 / t38;
  t487 = t79 * t79;
  t488 = t150 * t487;
  t504 =
    2.0 * E * t403 + 2.0 * t101 * d2rdt2 + d3rdt3 * t38 +
    2.0 * d2rdt2 * t79 + drdt * t445;
  t509 = (1.0 / 8.0 * i) * nmu;
  t510 = t509 * t11;
  t511 = t510 * t231;
  t518 = t109 * t79;
  t527 = t159 * t53 * t56;
  t528 = t527 * t182;
  t537 = t302 * t231;
  t538 = t479 * t110;
  t541 = t181 * d2thdt2;
  t545 = t312 * t110;
  t548 =
    t311 * t234 * t458 / 4.0 +
    t462 * t11 * t271 * t131 * t466 * t257 * t352 +
    t302 * t85 * t358 * t472 / 4.0 + t335 * t176 * t231 * t477 +
    t337 * t480 * t339 - t310 * t230 * t484 * t234 * t488 / 4.0 +
    t321 * t357 * t106 * t472 / 4.0 -
    t330 * t504 * t51 * t110 / 8.0 +
    t511 * t234 * mm * d2phidt2 * t150 +
    t366 * t230 * t44 * t466 * t257 * t518 -
    t330 * t106 * t159 * t110 / 4.0 + t372 * t528 / 4.0 +
    t311 * t480 * t188 / 2.0 + t317 * t480 * t204 / 4.0 -
    t537 * t538 / 4.0 + t372 * t121 * t541 / 8.0 - t537 * t545 / 4.0;
  t549 = t456 + t548;
  t552 = t381 * nmu;
  t558 = t552 * t11 * t231 * t234 * t274 * t151 / 8.0;
  t564 = t132 * t110;
  t570 = t118 * t43;
  t576 = t174 * mm;
  t577 = t257 * t109;
  t578 = t576 * t577;
  t580 = a * t12;
  t581 = t580 * t16;
  t582 = t19 * nmu;
  t583 = t582 * t100;
  t585 = t338 * t56;
  t590 = t26 * nmu;
  t591 = t590 * t100;
  t607 = t97 * t357;
  t615 = (-1.0 / 8.0 * i) * t163 * t127;
  t619 = nmu * t12;
  t620 = t16 * t100;
  t625 = t138 * t100;
  t626 = t625 * t173;
  t627 = t619 * t626;
  t628 = t57 * t46;
  t630 = 2.0 * t133 * t3 * dthdt;
  t633 = t85 * t100;
  t634 = t633 * t173;
  t635 = t619 * t634;
  t636 = t57 * t383;
  t639 = t176 * t176;
  t642 = t174 * t150;
  t645 = t256 * t376;
  t648 = t85 * t40;
  t649 = t648 * t173;
  t650 = t619 * t649;
  t652 = t79 * t3 * dthdt;
  t656 = t138 * t40;
  t659 = t57 * t353;
  t661 = t365 * t148;
  t663 = d2phidt2 * t56 * t109;
  t666 = t619 * t648;
  t670 = t619 * t633;
  t675 = (1.0 / 2.0 * i) * nmu;
  t676 = t675 * t12;
  t678 = t265 * dphidt;
  t679 = t678 * t204;
  t681 = (0.0 - i) * nmu;
  t683 = t633 * t51;
  t691 = t85 * t484;
  t696 =
    3.0 / 2.0 * t619 * t620 * t174 * t346 + t627 * t628 * t630 +
    t635 * t636 / 2.0 + nmu * t639 * t633 * t642 / 2.0 -
    t635 * t645 / 4.0 + t650 * t628 * t652 / 2.0 +
    t619 * t656 * t173 * t659 + t661 * t576 * t663 -
    t666 * t174 * t452 / 4.0 + t670 * t120 * t150 * t447 / 2.0 +
    t676 * t649 * t679 + t681 * t12 * t683 * t678 * t160 +
    t681 * t11 * t634 * t678 * t177 + t619 * t691 * t174 * t488 / 2.0;
  t697 = t310 * t634;
  t703 = t109 * t176;
  t707 = t174 * t331;
  t726 = t159 * t159;
  t731 = t676 * t634;
  t734 = t619 * t625;
  t738 = t678 * t188;
  t745 = t619 * t683;
  t747 = t57 * t541;
  t750 = t310 * t633;
  t755 =
    0.0 - t697 * t628 * t176 * t3 * dthdt -
    t310 * t649 * t57 * t703 * t79 + t670 * t707 / 4.0 -
    4.0 * t310 * t626 * t57 * t703 * t133 +
    2.0 * t310 * t683 * t57 * t703 * t159 -
    2.0 * t619 * t625 * t51 * t57 * t352 * t159 +
    t670 * t726 * t53 * t150 / 2.0 + t731 * t678 * t236 -
    t734 * t174 * t458 / 2.0 + (i) * t627 * t738 -
    t619 * t648 * t51 * t57 * t518 * t159 - t745 * t528 -
    t635 * t747 / 4.0 + t750 * t174 * t150 * t296 / 2.0;
  t758 = t83 * nmu;
  t760 = t466 * t160;
  t762 = t9 * nmu;
  t764 = t576 * t204;
  t766 = t762 * t148;
  t771 = t576 * t188;
  t773 = a * nmu;
  t774 = t773 * t148;
  t780 = t576 * t177;
  t787 = t163 / t15 / t14;
  t788 = t787 * t200;
  t792 =
    t229 * t241 * t243 / 4.0 + t251 +
    t90 * t87 * (t252 * nmu * t98 * t132 * t256 * t257 -
		 t261 * t263 * t267 / 8.0 +
		 t270 * t272 * t234 * t275 / 4.0 +
		 t270 * t241 * t234 * t280 / 8.0 -
		 t284 * t108 * t266 / 8.0 -
		 t284 * t191 * t266 / 8.0 + t17 * t29 * t549 +
		 t558) / 2.0 - t96 * t263 * t564 / 4.0 +
    t229 * t272 * t304 / 2.0 + t117 * t570 * t527 +
    (1.0 / 4.0 * i) * t95 * nmu * t607 * t477 +
    t581 * t583 * t234 * t585 / 4.0 -
    t197 * t198 * t591 * t183 / 8.0 +
    t114 * t11 * t21 * t176 * t100 * t43 * t121 +
    (1.0 / 4.0 * i) * t145 * nmu * t148 * t578 +
    (-1.0 / 4.0 * i) * t167 * nmu * t148 * t578 +
    t168 * t171 * t178 - t168 * t186 * t189 +
    t615 * t200 * t225 * t339 - t7 * (t8 * (t696 + t755) +
				      t758 * t148 * t760 +
				      t762 * t208 * t764 +
				      t766 * t576 * t236 +
				      t124 * nmu * t186 * t771 +
				      t774 * t174 * t255 * t577 /
				      4.0 +
				      t758 * t171 * t780) / 2.0 -
    t788 * t225 * t188 / 4.0;
  t795 = t7 * t87;
  t797 = t537 * t299 / 8.0;
  t799 = t311 * t304 / 4.0;
  t801 = t317 * t243 / 8.0;
  t803 = t330 * t545 / 8.0;
  t805 = t330 * t538 / 8.0;
  t806 = t511 * t477;
  t808 = t372 * t472 / 8.0;
  t809 = t799 - t797 + t801 - t803 - t805 + t806 + t808;
  t814 = t17 * t29 * t809 - t284 * t267 / 8.0;
  t816 = t795 * t814 / 2.0;
  t817 = t87 * t814;
  t819 = t97 * t817 * rp;
  t822 = t57 * rp;
  t824 = t17 * t12 * t129 * t132 * t822;
  t825 = t115 * nmu;
  t826 = t114 * t825;
  t828 = t826 * t131 * t121;
  t830 = t11 * t100;
  t833 = t114 * t21 * t830 * t43 * t121;
  t836 = (-2.0 * i) * a;
  t839 = t87 * rp;
  t840 = t138 * t29 * t839;
  t842 = t7 * t91;
  t843 = t842 * t87;
  t848 =
    (i) * a * t11 * t88 + t836 * t12 * t840 - t843 / 4.0 +
    t97 * t92 * rp / 2.0;
  t849 = t848 * nmu;
  t852 = t849 * t98 * t564 / 4.0;
  t862 = t57 * t182;
  t865 =
    t750 * t178 / 2.0 - t734 * t189 / 2.0 - t666 * t210 / 4.0 +
    t670 * t161 / 2.0 + t661 * t578 - t635 * t862 / 4.0;
  t866 = t8 * t865;
  t867 = t619 * t85;
  t869 = t274 * t109;
  t870 = t225 * t869;
  t871 = t114 * t867 * t870;
  t872 = t866 + t871;
  t877 = t11 * t138;
  t878 = t877 * t140;
  t884 = 2.0 * t19 * t142;
  t887 = t85 * t19;
  t888 = t887 * t142;
  t889 = t888 / 2.0;
  t892 = 2.0 * t877 * t143 * rp;
  t894 = t97 * t142 / 2.0;
  t896 = t97 * t19 / 2.0;
  t897 =
    0.0 - t878 + 3.0 * t115 * t140 * rp - t139 * t26 / 2.0 -
    t139 * t884 / 2.0 + t889 - t892 + t894 + t896;
  t898 = t897 * nmu;
  t900 = t224 * t110;
  t902 = t898 * t147 * t900 / 2.0;
  t903 = t163 * t16;
  t909 = 2.0 * t11 * t142;
  t912 =
    0.0 - t878 + 3.0 * t903 * t165 * rp - t164 * t26 / 2.0 -
    t164 * t909 / 2.0 + t889 - t892 + t894 + t896;
  t913 = t912 * nmu;
  t916 = t913 * t147 * t900 / 2.0;
  t917 = t128 * t200;
  t919 = t917 * t900 / 4.0;
  t920 = t150 * rp;
  t923 = 3.0 / 4.0 * t788 * t225 * t920;
  t925 = t223 * t900 / 8.0;
  t928 = 2.0 * t197 * t198 * t142 * nmu;
  t930 = t928 * t900 / 8.0;
  t931 =
    t816 - t819 + t824 + t828 + t833 - t852 + t85 * t872 * rp -
    t7 * rp * t865 - t902 + t916 + t919 - t923 + t925 + t930;
  t933 = (-1.0 / 8.0 * i) * nmu;
  t936 = t234 * t869;
  t940 = t234 * t274 * t352;
  t944 = t234 * t274 * t518;
  t947 = t933 * t11 * t231;
  t958 =
    t933 * t176 * t231 * t936 + t336 * t272 * t940 +
    t510 * t241 * t944 + t947 * t313 * t869 + t947 * t480 * t869 -
    t372 * t373 * t577 / 8.0 + t510 * t367 * t466 * t236;
  t961 = (-1.0 / 8.0 * i) * t46;
  t964 = t256 * t56;
  t977 = t265 * t150;
  t978 = t132 * t977;
  t988 = t256 * t577;
  t993 = t266 * t182;
  t998 = t255 * t56 * t109;
  t1012 = (1.0 / 8.0 * i) * t163 * t127;
  t1021 = t90 * t817 / 2.0;
  t1023 = t117 * t570 * t121;
  t1025 = t99 * t564 / 4.0;
  t1030 = t146 * t147 * t900 / 2.0;
  t1033 = t168 * t147 * t900 / 2.0;
  t1035 = t201 * t900 / 8.0;
  t1036 = t1021 + t1023 - t1025 - t7 * t872 / 2.0 - t1030 + t1033 + t1035;
  t1046 = t146 * t98;
  t1053 = (-1.0 / 8.0 * i) * a;
  t1054 = t1053 * nmu;
  t1057 = t1054 * t98;
  t1060 = (1.0 / 8.0 * i) * a;
  t1061 = t1060 * nmu;
  t1067 = t1061 * t607;
  t1071 = t114 * nmu;
  t1075 = t773 * t11;
  t1076 = t1075 * t231;
  t1077 = t255 * dphidt;
  t1085 = t146 * t11;
  t1092 = t197 * t12;
  t1093 = t590 * t87;
  t1094 = t1092 * t1093;
  t1097 = t590 * t357;
  t1101 = t124 * t11;
  t1102 = t29 * t87;
  t1106 = t1101 * t887 * t1102 + (i) * t580 * t88;
  t1107 = t1106 * nmu;
  t1108 = t43 * t43;
  t1109 = t1108 * t40;
  t1111 = t85 * t53;
  t1115 = t1108 * t100;
  t1116 = t1107 * t1115;
  t1117 = t138 * t53;
  t1121 = t1115 * t85;
  t1127 = t150 * t106;
  t1135 = t1085 * t231;
  t1145 = t787 * t12;
  t1146 = t1145 * t1093;
  t1147 = t57 * t352;
  t1151 = t57 * t518;
  t1156 = t221 * nmu * t87;
  t1162 = nmu * t1108;
  t1163 = t1162 * t100;
  t1172 = t1162 * t633;
  t1175 = t109 * t291;
  t1176 = t57 * t1175;
  t1179 = nmu * t43;
  t1181 = t109 * t106;
  t1190 = t633 * t53;
  t1199 = t1162 * t648;
  t1204 = t1162 * t625;
  t1209 = t106 * t106;
  t1212 = t1111 * t150;
  t1215 = t131 * t85;
  t1220 = t1109 * t85;
  t1223 = t1115 * t138;
  t1226 = t933 * t1121;
  t1245 = t1179 * t633;
  t1263 =
    t335 * t1220 * t679 + t675 * t1223 * t738 +
    t1226 * t265 * t663 - t1172 * t645 / 8.0 +
    t335 * t1108 * t1190 * t585 * t182 +
    t1162 * t648 * t53 * t235 * t652 / 4.0 +
    3.0 / 4.0 * t1162 * t620 * t57 * t109 * t345 +
    t1245 * t57 * t109 * t504 / 4.0 +
    t1162 * t691 * t57 * t109 * t487 / 4.0 +
    t1162 * t656 * t659 / 2.0 + t1162 * t625 * t53 * t235 * t630 / 2.0;
  t1276 = t261 * t1115;
  t1285 = t56 * t3;
  t1289 = t552 * t1121 * t265 * t1285 * dthdt / 8.0;
  t1294 =
    0.0 - t1107 * t131 * t1111 * t1127 / 4.0 - t1046 * t111 / 4.0 +
    t1094 * t111 / 8.0 + t1135 * t237 / 4.0 +
    t197 * t11 * t590 * t176 * t358 * t120 * t150 / 8.0 -
    t1146 * t132 * t1147 / 4.0 - t1094 * t52 * t1151 / 8.0 +
    t615 * t1156 * t131 * t120 * t339 +
    (1.0 / 8.0 * i) * t1106 * t1163 * t1111 * mm * t577 +
    (1.0 / 8.0 * i) * t145 * nmu * t607 * t477 -
    t90 * t87 * (t17 * t29 *
		 (t1172 * t636 / 4.0 + t1172 * t1176 / 8.0 -
		  2.0 * t1179 * t625 * t57 * t1181 * t133 -
		  t1179 * t648 * t57 * t1181 * t79 / 2.0 -
		  t1179 * t1190 * t235 * t106 * t3 * dthdt / 2.0 -
		  t1172 * t747 / 8.0 -
		  t1199 * t57 * t109 * t445 / 8.0 -
		  t1204 * t57 * t109 * t457 / 2.0 +
		  nmu * t1209 * t100 * t1212 / 4.0 +
		  t462 * t1215 * t678 * t1127 + t1263) +
		 t261 * t131 * t1111 * t274 * t106 / 4.0 -
		 t261 * t1109 * t1111 * t280 / 8.0 -
		 t1276 * t1117 * t275 / 4.0 +
		 t961 * t1163 * t1111 * t1077 * t56 - t1289) / 2.0;
  t1300 = t8 * t809 + t1057 * t978;
  t1302 = t67 * t29;
  t1303 = t85 * t1300 * t1302;
  t1304 = t877 * t19;
  t1305 = t26 * a;
  t1306 = t1305 * t29;
  t1309 = t3 * t29;
  t1310 = t26 * t2 * t1309;
  t1313 = (-1.0 / 2.0 * i) * t12;
  t1316 = t124 * t29;
  t1317 = t1316 * t888;
  t1321 = 2.0 * t1304 * t142 * t2 * t1309;
  t1322 = (1.0 / 2.0 * i) * t11;
  t1324 = a * t29;
  t1326 = t1322 * t85 * t1324 * t142;
  t1327 =
    (i) * t1304 * t1306 - 3.0 * t116 * t1310 +
    t1313 * t138 * t1306 + t1317 + t1321 + t1326;
  t1328 = t1327 * nmu;
  t1331 = t1328 * t98 * t564 / 4.0;
  t1335 = t56 * a;
  t1338 = (1.0 / 4.0 * i) * t19 * t127 * t222 * t358 * t120 * t1335;
  t1339 = t57 * t67;
  t1342 = 3.0 / 4.0 * t1146 * t132 * t1339;
  t1343 = t1305 * nmu;
  t1345 = t11 * t87;
  t1346 = t1345 * t100;
  t1349 = t615 * t1343 * t1346 * t233 * t57;
  t1350 = t83 * t29;
  t1353 = t1245 * t57 * t1181 / 4.0;
  t1355 = t1199 * t1151 / 8.0;
  t1357 = t1204 * t1147 / 4.0;
  t1359 = t1226 * t265 * t577;
  t1361 = t1172 * t862 / 8.0;
  t1362 = t1353 - t1355 - t1357 + t1359 - t1361;
  t1364 = t17 * t29 * t1362;
  t1367 = t1276 * t1111 * t274 / 8.0;
  t1368 = t1364 + t1367;
  t1375 = t17 * t3 * t1362;
  t1380 = t552 * t1115 * t1111 * t274 * t3 / 8.0;
  t1388 = t2 * a;
  t1389 = (-2.0 * i) * t1388;
  t1390 = t1389 * t877;
  t1391 = t19 * t30;
  t1392 = t87 * t3;
  t1395 = t2 * t11;
  t1398 = t1395 * t85 * t30 * t87;
  t1405 = t138 * t30 * t1392;
  t1407 = t85 * t3;
  t1408 = t1407 * t87;
  t1412 =
    (0.0 - t31 * t887 * t87 / 2.0 + t1390 * t1391 * t1392 +
     5.0 / 2.0 * t1398 + t1101 * t887 * t1392 +
     (4.0 * i) * t1388 * t12 * t1405 + (i) * t580 * t1408) * nmu * t1115;
  t1414 = t1412 * t1212 / 8.0;
  t1415 = (0.0 - i) * t109;
  t1453 = t1111 * t869;
  mgv1 =
    t17 * t29 * (0.0 - t1046 * t192 / 4.0 - t146 * t263 * t564 / 4.0 +
		 t7 * (t8 * t549 + t1054 * t263 * t978 +
		       t1057 * t108 * t977 +
		       t1061 * t97 * t322 * t944 +
		       t1057 * t191 * t977 +
		       t1067 * t233 * t265 * t236 +
		       t1071 * t877 * t357 * t940 -
		       t1076 * t234 * t1077 * t150 / 8.0) / 2.0 +
		 t1085 * t272 * t304 / 2.0 +
		 t1085 * t241 * t243 / 4.0 + t1094 * t192 / 8.0 -
		 t1092 * t1097 * t237 / 8.0 +
		 t1107 * t1109 * t1111 * t204 / 8.0 +
		 t1116 * t1117 * t188 / 4.0 +
		 t1107 * t1121 * t862 / 8.0 + t1294) + t1303 -
    t1331 + t1338 + t1342;
  t1457 =
    mgv1 + t1349 + t1350 * t795 * t1368 -
    t98 * t1368 * t2 * t1309 - t90 * t87 * (t1375 - t1380) / 2.0 -
    t1414 +
    t1415 * (t7 * (t8 * t958 + t1076 * t234 * t998 / 8.0) / 2.0 +
	     (-1.0 / 8.0 * i) * t145 * nmu * t98 * t978 +
	     t1012 * t1156 * t978 -
	     t90 * t87 * (t17 * t29 *
			  (t335 * t1215 * t265 * t1127 +
			   t933 * t1220 * t265 * t204 +
			   t365 * t1223 * t265 * t188 +
			   t1172 * t988 / 8.0 + t1226 * t993) +
			  t252 * t1162 * t633 * t964) / 2.0 +
	     (-1.0 / 8.0 * i) * t1106 * t1163 * t1453);
  t1461 = t124 * t97;
  t1462 = t19 * t29;
  t1464 = t7 * t1300 / 2.0;
  t1466 = t1046 * t564 / 4.0;
  t1468 = t1094 * t564 / 8.0;
  t1473 = t1116 * t1212 / 8.0;
  t1474 = t1464 - t1466 + t1468 - t90 * t87 * t1368 / 2.0 - t1473;
  t1475 = t87 * t1474;
  t1483 = (2.0 * i) * a;
  t1484 = t1483 * t12;
  t1489 = t173 * dthdt;
  t1490 = t1489 * t110;
  t1493 = t233 * dthdt;
  t1494 = t1493 * t110;
  t1499 = t224 * dthdt;
  t1503 = dthdt * t53;
  t1504 = t1503 * t56;
  t1505 = t132 * t1504;
  t1507 = t51 * dthdt;
  t1508 = t1507 * t53;
  t1523 = t235 * t3;
  t1537 =
    t368 * t1508 * t339 - t324 * t1508 * t204 / 4.0 -
    t359 * t1508 * t188 / 2.0 + t330 * t312 * dthdt * t110 / 4.0 -
    t372 * t51 * t291 * t53 * t1523 / 4.0 + t537 * t1494 / 4.0 +
    t330 * t233 * d2thdt2 * t110 / 8.0 + t330 * t479 * dthdt * t110 / 4.0;
  t1540 = t270 * t231;
  t1541 = t1493 * t266;
  t1544 =
    t17 * t29 * t1537 - t797 + t799 + t801 - t803 - t805 + t808 +
    t1540 * t1541 / 8.0 + t806;
  t1550 = t1503 * t188;
  t1553 = t1503 * mm * t577;
  t1556 = t291 * t53 * t1523;
  t1565 = t1503 * t204;
  t1570 = t1489 * t53;
  t1571 = t1570 * t869;
  t1573 =
    t8 * (0.0 - t697 * t1503 * t177 + t627 * t1550 + t731 * t1553 +
	  t635 * t1556 / 2.0 - t745 * t1503 * t160 -
	  t670 * t173 * d2thdt2 * t110 / 4.0 +
	  t650 * t1565 / 2.0) + t766 * t1571;
  t1581 = t1057 * t1505 - t330 * t299 / 8.0;
  t1582 = t795 * t1581;
  t1584 = t87 * t1581;
  t1585 = t1584 * rp;
  t1587 = t16 * t8;
  t1588 = t619 * t100;
  t1589 = t1587 * t1588;
  t1598 =
    t26 * (t1582 / 2.0 - t97 * t1585 -
	   t1589 * t1570 * t920 / 4.0 + t138 * rp * t1588 * t1490 / 4.0);
  t1602 = t1076 * t1541 / 8.0 + t947 * t936;
  t1606 = (1.0 / 8.0 * i) * t138;
  t1612 =
    t8 * (0.0 - t169 * t1490 / 2.0 + t232 * t1494 / 4.0 +
	  t149 * t1490 / 2.0 - t201 * t1499 * t110 / 8.0 +
	  t22 * t1505 + t90 * t87 * t1544 / 2.0 -
	  t7 * t1573 / 2.0) - t1598 +
    a * (t90 * t87 * t1602 / 2.0 + t1606 * t8 * t1588 * t1571);
  t1617 = t138 * t8;
  t1618 = t1617 * t1588;
  t1621 = t90 * t1584 / 2.0 + t1618 * t1490 / 8.0;
  t1632 = t1493 * t977;
  t1634 = t8 * t1537 + t1067 * t1632;
  t1637 = t85 * dthdt;
  t1638 = t1637 * t110;
  t1651 = t335 * t1121;
  t1658 =
    t1199 * t1565 / 4.0 - t1172 * d2thdt2 * t53 * t150 / 8.0 +
    t1204 * t1550 / 2.0 + t1651 * t1553 + t1172 * t1556 / 4.0 -
    t1245 * t1503 * t1127 / 2.0;
  t1661 = t1637 * t266;
  t1664 =
    t17 * t29 * t1658 + t1353 - t1355 - t1357 + t1359 - t1361 -
    t1276 * t1661 / 8.0;
  t1668 = t1503 * t150;
  t1669 = t132 * t1668;
  t1672 =
    t7 * t1634 / 2.0 + t1116 * t1638 / 8.0 + t1135 * t1494 / 4.0 -
    t90 * t87 * t1664 / 2.0 - t1094 * t1669 / 8.0;
  t1675 = t1587 * nmu;
  t1676 = t1675 * t1346;
  t1677 = t1493 * t1339;
  t1679 = t1676 * t1677 / 8.0;
  t1681 = t633 * t1504;
  t1685 = t1060 * t1162 * t1681 + t1163 * t1212 / 8.0;
  t1686 = t795 * t1685;
  t1687 = t1350 * t1686;
  t1688 = t1685 * t2;
  t1690 = t98 * t1688 * t1309;
  t1691 = t3 * nmu;
  t1697 = t1060 * t1691 * t1108 * t633 * dthdt * t110 + t1364 + t1367;
  t1701 = (1.0 / 16.0 * i) * t138;
  t1703 = t310 * t357;
  t1711 = 0.0 - t773 * t1115 * t1661 / 8.0 + t509 * t1115 * t1453;
  t1715 = t1701 * t8 * t1703 * t1632 - t90 * t87 * t1711 / 2.0;
  t1717 =
    t17 * t29 * t1672 + t1679 + t1464 - t1466 + t1468 + t1687 -
    t1690 - t90 * t87 * t1697 / 2.0 - t1473 + t1415 * t1715;
  t1722 = t1617 * nmu * t1346;
  t1724 = t1722 * t1494 / 16.0;
  t1727 = t90 * t87 * t1685 / 2.0;
  t1728 = t1724 - t1727;
  t1729 = t87 * t1728;
  t1743 = t26 * t1621;
  t1744 = t1324 * t1743;
  t1750 = (1.0 / 2.0 * i) * t163 * t138;
  t1755 = (-1.0 / 16.0 * i) * t11;
  t1757 = t87 * a;
  t1758 = t1757 * t3;
  t1759 = t1755 * t138 * t1758;
  t1761 = t1759 * t1163 * t1668;
  t1762 = t1679 + t1687 - t1690 + t1761;
  t1763 = t87 * t1762;
  t1772 = t91 * t91;
  t1774 = (-1.0 - t1772) * t87;
  t1794 = t274 * t67;
  t1802 = t877 * t87;
  t1803 = t10 * nmu;
  t1812 = t127 * t8 * nmu;
  t1813 = t1345 * t131;
  t1814 = t1812 * t1813;
  t1815 = t56 * t29;
  t1816 = t2 * t2;
  t1817 = t1816 * t4;
  t1821 = t1814 * t1508 * t1815 * t1817 / 2.0;
  t1822 = t29 * t2;
  t1826 = t1676 * t1493 * t57 * t1822 / 8.0;
  t1827 = t1675 * t1813;
  t1831 = t1827 * t1508 * t150 * t5 / 8.0;
  t1832 = (2.0 * i) * t1388;
  t1833 = t1832 * t30;
  t1836 = t1833 * t230 * t1685 * t3;
  t1839 =
    t1415 * ((1.0 / 8.0 * i) * t16 * t8 * t1703 * t233 * t1503 *
	     t1794 + t1350 * t795 * t1711 -
	     t98 * t1711 * t2 * t1309 +
	     t1802 * t1803 * t1115 * dthdt * t977 / 16.0) +
    t1338 + t1821 - t1826 + t1831 + t1303 - t1331 + t1836 +
    t1342 + t17 * t3 * t1672 + t1349;
  t1842 = t2 * t138;
  t1844 = t1842 * t1392 * nmu;
  t1846 = t1844 * t1115 * t1504;
  t1848 = t83 * t3;
  t1849 = t1848 * t1686;
  t1851 = t4 * t30;
  t1854 = 4.0 * t1802 * t1685 * t1816 * t1851;
  t1858 = t98 * t1688 * t30;
  t1860 = t98 * t1688 * t4;
  t1861 = (-1.0 / 8.0 * i) * t11;
  t1862 = t1861 * t16;
  t1863 = t87 * t1388;
  t1864 = t1863 * t4;
  t1866 = t1163 * t1504;
  t1871 = t1053 * t1162;
  t1897 = (-1.0 / 4.0 * i) * t19 * t127;
  t1902 = t1145 * t1097;
  t1913 =
    t1350 * t795 * t1697 - t1846 / 16.0 + t1849 - t1854 +
    (i) * t46 * t1715 * t3 + t1858 - t1860 +
    t1862 * t1864 * t1866 - t98 * t1697 * t2 * t1309 -
    t90 * t87 * (t1871 * t1681 + t1375 - t1380) / 2.0 - t1414 +
    t17 * t29 * (t85 * t1634 * t1302 + t1412 * t1638 / 8.0 +
		 t1328 * t11 * t231 * t1494 / 4.0 +
		 t1350 * t795 * t1664 - t98 * t1664 * t2 * t1309 -
		 t90 * t87 * (t17 * t3 * t1658 + t1289) / 2.0 +
		 t1897 * t1156 * t132 * t1503 * t1335 -
		 3.0 / 4.0 * t1902 * t1677 +
		 t1012 * t1305 * t310 * t358 * t1507 * t57);
  t1920 = t30 * t87;
  t1926 = t1621 * t2 * t1309;
  t1929 =
    (-2.0 * i) * t12 * t138 * t1744 + t9 * t29 * t842 * t1729 +
    t1750 * t1744 + t98 * t1717 * t2 * t1309 +
    t1461 * t1462 * t1763 + t1390 * t1391 * t1729 * t3 +
    t1461 * t19 * t3 * t1729 + t90 * t1774 * t1728 / 4.0 -
    t31 * t85 * t19 * t87 * t1728 / 2.0 +
    t90 * t92 * t1762 / 4.0 + t85 * t1612 * t1302 +
    t1316 * t795 * t1717 + t90 * t87 * (t1839 + t1913) / 2.0 +
    (8.0 * i) * t1388 * t139 * t1920 * t1728 * t3 -
    12.0 * t115 * t140 * t1926;
  t1938 = (-1.0 / 8.0 * i) * t12;
  t1944 = t1570 * t56 * t2 * t3;
  t1947 =
    t1316 * t1582 + t98 * t1581 * t2 * t1309 +
    t1938 * t138 * t1803 * t1669 + t1589 * t1944 / 4.0;
  t1963 = t903 * t11;
  t1967 =
    (0.0 - i) * t11 * t138 * t140 * t1324 - 3.0 * t1963 * t1310 +
    t1750 * t1306 + t1317 + t1321 + t1326;
  t1977 = t97 * t91;
  t1984 =
    t1398 + t1832 * t12 * t1405 + t84 * t1408 +
    t114 * t29 * t843 - t1977 * t87 * t2 * t1309 / 2.0 - t90 * t1774 / 4.0;
  t1998 = t787 * t198 * t591;
  t2003 = t12 * t100 * t173;
  t2036 =
    0.0 - t1967 * nmu * t148 * t1490 / 2.0 +
    t1984 * nmu * t11 * t231 * t1494 / 4.0 +
    t1328 * t148 * t1490 / 2.0 + t1897 * t200 * t1499 * t57 * a -
    3.0 / 4.0 * t1998 * t1944 + t1012 * t1343 * t2003 * t1504 -
    t1822 * t16 * t582 * t11 * t1505 / 4.0 +
    (0.0 - i) * t1388 * t12 * t128 * t29 * nmu * t250 +
    t2 * t12 * t16 * t29 * nmu * t1505 / 4.0 + t251 +
    t1316 * t795 * t1544 + t98 * t1544 * t2 * t1309 +
    t90 * t87 * (t17 * t3 * t1537 - t558) / 2.0 - t85 * t1573 * t1302;
  t2040 = (-1.0 / 8.0 * i) * t138;
  t2042 = t1691 * t11;
  t2045 = t1324 * t85;
  t2048 = rp * t2;
  t2049 = t2048 * t1309;
  t2052 = (1.0 / 4.0 * i) * t12;
  t2053 = t2052 * t16;
  t2054 = t10 * t118;
  t2057 = t57 * t109 * rp;
  t2060 = t1812 * t2003;
  t2069 =
    t26 * (t230 * t1581 * t1302 + t2040 * a * t2042 * t1669 +
	   (i) * t2045 * t1585 - 4.0 * t877 * t1584 * t2049 +
	   t2053 * t2054 * t1493 * t2057 -
	   t2060 * t1504 * t2048 * t3 + t16 * rp * t1588 * t1944 / 2.0);
  t2111 =
    9.0 / 2.0 * t1395 * t85 * t1920 * t1728 -
    2.0 * t139 * t140 * t1947 - t164 * t165 * t1947 / 2.0 +
    t97 * t92 * t1728 * t2 * t1309 / 2.0 + t7 * (t8 * t2036 -
						 t2069 +
						 a * (t1316 *
						      t795 *
						      t1602 +
						      t98 *
						      t1602 * t2 *
						      t1309 +
						      t139 * a *
						      t1691 *
						      t100 *
						      t1632 /
						      8.0 +
						      (1.0 / 4.0 *
						       i) * t16 *
						      t8 * t1588 *
						      t1570 *
						      t1794)) /
    2.0 + 3.0 / 2.0 * t145 * t1947 - t1967 * t1621 -
    t167 * t1947 / 2.0 + 3.0 * t1327 * t1621 +
    3.0 * t1984 * t1728 + 3.0 * t95 * t1762 +
    t1484 * t1407 * t1729 + t1484 * t86 * t1763 -
    3.0 * t903 * t165 * t1926 + (3.0 * i) * t11 * t138 * t19 * t1744;
  t2114 = t7 * t8;
  t2116 = t57 * t291;
  t2117 = t132 * t2116;
  t2121 = t1061 * t98 * t2117 + t330 * t1494 / 8.0;
  t2127 = t90 * t87 * t2121 / 2.0 - t1618 * t707 / 8.0;
  t2133 = t1871 * t633 * t2116 - t1172 * t1668 / 8.0;
  t2139 = 0.0 - t90 * t87 * t2133 / 2.0 - t1722 * t332 / 16.0;
  t2142 = t17 * t29 * t2139 + t1724 - t1727;
  t2147 = t85 * t8;
  t2150 = t795 * t2121;
  t2152 = t2121 * t2;
  t2155 = (1.0 / 8.0 * i) * t12;
  t2160 = t291 * t2;
  t2165 =
    t1316 * t2150 + t98 * t2152 * t1309 +
    t2155 * t138 * t1803 * t132 * t1176 - t1589 * t180 * t2160 * t3 / 4.0;
  t2168 = t795 * t2142;
  t2170 = t2142 * t2;
  t2175 = t795 * t2133;
  t2177 = t2133 * t2;
  t2180 = (1.0 / 16.0 * i) * t11;
  t2190 =
    t1350 * t2175 - t98 * t2177 * t1309 +
    t2180 * t138 * t1758 * t1163 * t1176 -
    t1676 * t234 * t56 * t291 * t67 / 8.0;
  t2193 =
    t17 * t3 * t2139 + t17 * t29 * t2190 + t1679 + t1687 - t1690 + t1761;
  t2207 = t2127 * t2;
  t2212 = t124 * t3;
  t2214 = t1389 * t30;
  t2235 = t138 * a;
  t2236 = t2235 * nmu;
  t2237 = t1938 * t2236;
  t2244 = t29 * t291;
  t2253 =
    t2212 * t2150 + t2214 * t230 * t2121 * t3 +
    t1842 * t2042 * t2117 / 4.0 +
    4.0 * t1802 * t2121 * t1816 * t1851 +
    (1.0 / 2.0 * i) * t12 * t16 * t1388 * t4 * nmu * t2117 -
    t98 * t2152 * t30 + t98 * t2152 * t4 + t2237 * t2117 -
    t2060 * t57 * t29 * t291 * t1816 * t4 +
    t1589 * t180 * t2244 * t2 / 4.0 - t1675 * t2003 * t110 * t426 / 4.0;
  t2256 = t17 * t29;
  t2276 = (-1.0 / 4.0 * i) * t11 * t16;
  t2293 = t1163 * t2116;
  t2299 = t271 * a;
  t2300 = t1755 * t2299;
  t2314 =
    t1848 * t2175 + t1833 * t230 * t2133 * t3 +
    t1844 * t1115 * t2116 / 8.0 -
    4.0 * t1802 * t2133 * t1816 * t1851 +
    (1.0 / 4.0 * i) * t11 * t16 * t1864 * t2293 +
    t98 * t2177 * t30 - t98 * t2177 * t4 + t2300 * t2293 -
    t1814 * t121 * t2244 * t1817 / 2.0 +
    t1676 * t234 * t1815 * t2160 / 8.0 - t1827 * t121 * t1175 * t5 / 8.0;
  t2321 = t2180 * t2299;
  t2323 =
    t1849 + t2276 * t1864 * t1866 + t1836 + t17 * t29 * t2314 +
    t1821 - t1826 + t1831 + (i) * t1324 * t2139 +
    t836 * t3 * t2190 - t1846 / 8.0 + t2321 * t1866 - t1854 + t1858 - t1860;
  t2327 =
    4.0 * t1617 * t2127 * t1817 * t30 +
    2.0 * t2147 * t2165 * t1302 - t2147 * t2207 * t30 +
    t2147 * t2207 * t4 + t2114 * t2253 / 2.0 +
    t2256 * t795 * t2193 + t2212 * t2168 +
    t2214 * t230 * t2142 * t3 +
    4.0 * t1802 * t2142 * t1816 * t1851 +
    2.0 * t98 * t2193 * t2 * t1309 - t98 * t2170 * t30 +
    t98 * t2170 * t4 + t90 * t87 * t2323 / 2.0;
  t2335 = drdt * t53;
  t2337 = t2335 * mm * t577;
  t2341 = t2335 * t204;
  t2346 = t2335 * t56;
  t2350 = t2335 * t188;
  t2352 =
    0.0 - t670 * t173 * d2rdt2 * t110 / 4.0 + t731 * t2337 -
    t697 * t2335 * t177 + t650 * t2341 / 2.0 -
    t745 * t2335 * t160 + t635 * t2346 * t182 / 2.0 + t627 * t2350;
  t2363 = t590 * t148;
  t2372 = t173 * drdt;
  t2373 = t2372 * t53;
  t2374 = t2373 * t869;
  t2376 =
    t8 * t2352 - t590 * t171 * t178 / 2.0 +
    t590 * t186 * t189 / 2.0 + t590 * t208 * t210 / 4.0 -
    t2363 * t161 / 2.0 + t2363 * t183 / 4.0 +
    (1.0 / 4.0 * i) * t26 * nmu * t148 * t578 + t766 * t2374;
  t2379 = t233 * drdt;
  t2380 = t2379 * t266;
  t2384 = t51 * drdt * t53;
  t2388 = t2379 * t110;
  t2411 =
    0.0 - t372 * t2384 * t236 / 4.0 + t537 * t2388 / 4.0 -
    t359 * t2384 * t188 / 2.0 + t330 * t312 * drdt * t110 / 4.0 -
    t324 * t2384 * t204 / 4.0 + t330 * t479 * drdt * t110 / 4.0 +
    t368 * t2384 * t339 + t330 * t233 * d2rdt2 * t110 / 8.0;
  t2414 = t1540 * t2380 / 8.0 + t17 * t29 * t2411;
  t2415 = t87 * t2414;
  t2418 = t132 * t2346;
  t2421 = t224 * drdt * t110;
  t2426 = t2372 * t110;
  t2431 =
    0.0 - t7 * t2376 / 2.0 + t90 * t2415 / 2.0 + t22 * t2418 -
    t201 * t2421 / 8.0 + t232 * t2388 / 4.0 + t149 * t2426 / 2.0 -
    t169 * t2426 / 2.0;
  t2433 = t773 * t100;
  t2435 = t2379 * t822;
  t2436 = t2053 * t2433 * t2435;
  t2438 = t2040 * t1075 * t2418;
  t2439 = t8 * nmu;
  t2440 = t2439 * t148;
  t2444 = 0.0 - t2440 * t2426 - t590 * t147 * t900;
  t2445 = t85 * t2444 / 4.0;
  t2446 = t2445 * rp;
  t2447 = rp * nmu;
  t2448 = t2447 * t148;
  t2450 = t2448 * t2426 / 2.0;
  t2451 = 2.0 * t142 * nmu;
  t2454 = t2451 * t147 * t900 / 4.0;
  t2455 = 0.0 - t2450 + t866 - t2454 + t871;
  t2458 =
    t2436 + t1023 + t1021 + t2438 - t1025 + t2446 -
    t7 * t2455 / 2.0 - t1030 + t1033 + t1035;
  t2470 =
    (-1.0 / 4.0 * i) * t8 * nmu * t148 * t2374 +
    (-1.0 / 4.0 * i) * t26 * t867 * t870;
  t2475 =
    t8 * t2431 - t26 * t2458 + a * (t773 * t186 * t2380 / 8.0 -
				    t7 * t2470 / 2.0);
  t2479 = t7 * t2444 / 4.0;
  t2481 = t2237 * t2418 - t2479 / 2.0;
  t2485 = t140 * t2481;
  t2495 = t590 * t11;
  t2502 = t590 * t98;
  t2507 = t2495 * t231;
  t2510 = t2379 * t977;
  t2516 =
    t8 * t2411 + t590 * t263 * t564 / 8.0 -
    t2495 * t272 * t304 / 4.0 - t2495 * t241 * t243 / 8.0 +
    t2502 * t111 / 8.0 + t2502 * t192 / 8.0 - t2507 * t237 / 8.0 +
    t1067 * t2510 + (-1.0 / 8.0 * i) * t26 * nmu * t607 * t477;
  t2521 = t85 * drdt;
  t2522 = t2521 * t110;
  t2525 = t2335 * t150;
  t2526 = t132 * t2525;
  t2551 =
    t17 * t29 * (t1651 * t2337 + t1199 * t2341 / 4.0 -
		 t1172 * d2rdt2 * t53 * t150 / 8.0 -
		 t1245 * t2335 * t1127 / 2.0 +
		 t1204 * t2350 / 2.0 +
		 t1162 * t633 * drdt * t862 / 4.0) -
    t1276 * t2521 * t266 / 8.0;
  t2552 = t87 * t2551;
  t2559 = t2439 * t11 * t231;
  t2562 = t2559 * t2388 + t2502 * t564;
  t2563 = t85 * t2562 / 8.0;
  t2564 = t2563 * t1302;
  t2567 = t1115 * t2346;
  t2569 = t1822 * t271 * nmu * t2567 / 16.0;
  t2572 = t1115 * drdt;
  t2575 = t1862 * t1863 * nmu * t2572 * t57 * t1309;
  t2577 = t1759 * t1163 * t2525;
  t2586 =
    (1.0 / 8.0 * i) * t8 * nmu * t607 * t2510 +
    (1.0 / 8.0 * i) * t26 * nmu * t98 * t978;
  t2589 = t1757 * nmu;
  t2596 =
    t17 * t29 * (t7 * t2516 / 2.0 + t1135 * t2388 / 4.0 +
		 t1116 * t2522 / 8.0 - t1094 * t2526 / 8.0 -
		 t90 * t2552 / 2.0) + t2564 - t2569 + t2575 +
    t2577 + t1415 * (t7 * t2586 / 2.0 + t877 * t2589 * t2572 * t266 / 16.0);
  t2597 = t87 * t2596;
  t2601 = t7 * t2562 / 16.0;
  t2603 = t2300 * t1163 * t2346;
  t2604 = t2601 + t2603;
  t2605 = t87 * t2604;
  t2606 = t1462 * t2605;
  t2613 = t86 * t2605;
  t2618 = drdt * dthdt;
  t2622 = t330 * t2388;
  t2624 = t1071 * t98 * t132 * t2618 * t57 + t2622 / 8.0;
  t2625 = t87 * t2624;
  t2628 = t2372 * dthdt * t110;
  t2633 = t2440 * t2628 / 2.0 + t2363 * t1490 / 4.0;
  t2635 = t90 * t2625 - t7 * t2633;
  t2637 = t8 * t2635 / 2.0 - t1743;
  t2639 = t2379 * t1668;
  t2644 = 0.0 - t2559 * t2639 / 4.0 - t2507 * t1494 / 8.0;
  t2651 = t9 * t1163 * t2521 * t1504 - t1172 * t2525 / 8.0;
  t2652 = t87 * t2651;
  t2654 = t7 * t2644 - t90 * t2652;
  t2657 = t17 * t29 * t2654 / 2.0 + t2601 + t2603;
  t2658 = t87 * t2657;
  t2662 = t85 * t2637;
  t2664 = t795 * t2657;
  t2668 = t85 * t2644;
  t2670 = t795 * t2651;
  t2675 = (1.0 / 8.0 * i) * t11;
  t2677 = t1757 * t1691;
  t2679 = t2572 * t1668;
  t2684 =
    t2564 + t2575 + t17 * t3 * t2654 / 2.0 +
    t17 * t29 * (t2668 * t1302 + t1350 * t2670 -
		 t98 * t2651 * t2 * t1309 +
		 t2675 * t138 * t2677 * t2679) + t2577 - t2569;
  t2685 = t87 * t2684;
  t2691 = t795 * t2624;
  t2699 = t85 * t2633;
  t2701 =
    t1316 * t2691 + t98 * t2624 * t2 * t1309 +
    t2052 * t138 * t2054 * t2639 - t2699 * t1302;
  t2704 = t8 * t2701 - t26 * t1947;
  t2717 = t26 * t2481;
  t2722 = 0.0 - t2450 - t2454;
  t2725 = t2438 + t2436 + t2446 - t7 * t2722 / 2.0;
  t2746 = t2451 * t148;
  t2768 = t9 * t825;
  t2789 =
    t85 * t2376 * rp - t7 * (2.0 * rp * t2352 -
			     t2451 * t171 * t178 / 2.0 +
			     t2451 * t186 * t189 / 2.0 +
			     t2451 * t208 * t210 / 4.0 -
			     t2746 * t161 / 2.0 +
			     t2746 * t183 / 4.0 +
			     (1.0 / 2.0 * i) * t142 * nmu * t148 *
			     t578) / 2.0 + t795 * t2414 / 2.0 -
    t97 * t2415 * rp + t9 * t11 * t21 * t2418 +
    (i) * t580 * t127 * t583 * t2435 + t2768 * t2418 -
    t917 * t2421 / 4.0 + 3.0 / 4.0 * t1998 * t2373 * t920 -
    t223 * t2421 / 8.0 - t928 * t2421 / 8.0 +
    t849 * t11 * t231 * t2388 / 4.0 + t898 * t148 * t2426 / 2.0 -
    t913 * t148 * t2426 / 2.0;
  t2794 = t310 * t100;
  t2796 = (1.0 / 2.0 * i) * t16 * a * t2794 * t2435;
  t2797 = t826 * t2418;
  t2801 = t57 * t1;
  t2803 = (0.0 - i) * t12 * t127 * t2433 * t2379 * t2801;
  t2806 = t138 * t2444 * t1;
  t2808 = t85 * t2722 * rp;
  t2811 = t670 * t2426;
  t2815 = t670 * t642;
  t2820 =
    t828 + t833 + t2796 + t2797 + t824 + t2803 + t816 - t819 -
    t852 - t2806 + t2808 + t2445 + t85 * t2455 * rp -
    t7 * (0.0 - t2811 / 2.0 + 2.0 * rp * t865 - t2815 / 2.0) / 2.0 -
    t902 + t916 + t919 - t923 + t925 + t930;
  t2826 = t274 * rp;
  t2850 = t2717 * rp;
  t2858 =
    0.0 - t85 * t2475 * rp + t795 * t2596 / 2.0 - 5.0 * t877 * t2485 -
    t164 * t2717 / 2.0 - t97 * t2597 * rp - t167 * t2725 / 2.0 -
    2.0 * t139 * t2717 + 3.0 * t897 * t2481 +
    t7 * (2.0 * rp * t2431 + t8 * t2789 - 2.0 * t142 * t2458 -
	  t26 * t2820 + a * (t2235 * t2794 * t2380 / 8.0 -
			     t581 * t570 * t2384 * t2826 / 4.0 +
			     t85 * t2470 * rp -
			     t7 * ((-1.0 / 2.0 * i) * rp * nmu *
				   t148 * t2374 +
				   (-1.0 / 2.0 * i) * t142 *
				   t867 * t870) / 2.0)) / 2.0 -
    t912 * t2481 + 3.0 / 2.0 * t145 * t2725 +
    12.0 * t116 * t2850 + 3.0 * t848 * t2604 - t164 * t165 * t2725 / 2.0;
  t2869 = t2451 * t11;
  t2876 = t2451 * t98;
  t2881 = t2869 * t231;
  t2895 = t124 * t85;
  t2898 = t1483 * t877;
  t2937 = t1309 * rp;
  t2939 = t138 * t2562 * t2 * t2937 / 2.0;
  t2941 = t2447 * t11 * t231;
  t2946 = t2941 * t2388 / 4.0 + t2876 * t564 / 8.0;
  t2948 = t85 * t2946 * t1302;
  t2951 = t2572 * t822;
  t2953 = t1822 * t342 * nmu * t2951 / 8.0;
  t2959 = t100 * drdt;
  t2960 = t2959 * t53;
  t2963 =
    (-1.0 / 8.0 * i) * t16 * t87 * t1388 * nmu * t1108 * t2960 * t1285 * t29;
  t2964 = t1322 * t127;
  t2970 = t2964 * t1863 * t1162 * t2960 * t1285 * t29 * rp;
  t2971 = (-1.0 / 16.0 * i) * t138;
  t2975 = t2971 * t87 * t1803 * t2572 * t110;
  t2976 = t2675 * t16;
  t2979 = t2976 * t2677 * t2572 * t2057;
  t2993 = t773 * t1108;
  t3013 = t2563 * rp;
  t3015 = t7 * t2946 / 2.0;
  t3017 = t2971 * t2589 * t2567;
  t3019 = t2976 * t2589 * t2951;
  t3020 = 0.0 - t3013 + t3015 + t3017 + t3019;
  t3021 = t87 * t3020;
  t3024 = t2605 * rp;
  mgv3 = 0.0 - t164 * t909 * t2481 / 2.0;
  mgv4 =
    t90 * t87 * (t17 * t29 *
		 (0.0 - t85 * t2516 * rp +
		  t7 * (2.0 * rp * t2411 +
			t2451 * t263 * t564 / 8.0 -
			t2869 * t272 * t304 / 4.0 -
			t2869 * t241 * t243 / 8.0 +
			t2876 * t111 / 8.0 + t2876 * t192 / 8.0 -
			t2881 * t237 / 8.0 +
			(-1.0 / 4.0 * i) * t142 * nmu * t607 *
			t477) / 2.0 +
		  t898 * t11 * t231 * t2388 / 4.0 +
		  (t2895 * t1462 * t87 + t2898 * t1462 * t839 +
		   (3.0 / 2.0 * i) * a * t11 * t88 +
		   (-4.0 * i) * a * t12 * t840) * nmu * t1115 *
		  t2522 / 8.0 - t128 * t12 * t1093 * t2526 / 4.0 +
		  3.0 / 4.0 * t1902 * t2379 * t2057 -
		  t197 * t26 * t310 * t87 * t2526 / 8.0 -
		  t1092 * t2451 * t87 * t2526 / 8.0 -
		  t795 * t2551 / 2.0 + t97 * t2552 * rp) - t2939 +
		 t2948 + t2953 + t2963 + t2970 + t2975 + t2979 +
		 t1415 * (0.0 - t85 * t2586 * rp +
			  t7 * ((1.0 / 4.0 * i) * rp * nmu *
				t607 * t2510 +
				(1.0 / 4.0 * i) * t142 * nmu *
				t98 * t978) / 2.0 +
			  t271 * t2993 * t2959 * t266 / 16.0 -
			  t11 * t16 * t87 * t2993 * t2960 *
			  t2826 / 8.0)) / 2.0 + 3.0 * t1963 * t2850;
  mgv2 = mgv3 + mgv4;
  mgv1 =
    mgv2 + t2895 * t2606 + t1461 * t1462 * t3021 +
    t2898 * t1462 * t3024 - 2.0 * t139 * t884 * t2481;
  t3052 =
    mgv1 - 2.0 * t139 * t140 * t2725 +
    (-8.0 * i) * t12 * t2235 * t1102 * t2604 * rp +
    t842 * t2605 / 4.0 + t90 * t92 * t3020 / 4.0 -
    t1977 * t3024 / 2.0 + t1484 * t86 * t3021 +
    (7.0 / 2.0 * i) * a * t11 * t2613 + 3.0 * t95 * t3020;
  t3059 = t2625 * rp;
  t3065 = t2448 * t2628 + t2746 * t1490 / 4.0;
  t3071 =
    rp * t2635 + t8 * (t2691 / 2.0 - t97 * t3059 + t2699 * rp -
		       t7 * t3065 / 2.0) - 2.0 * t142 * t1621 - t1598;
  t3075 = t2658 * rp;
  t3082 = 0.0 - t2941 * t2639 / 2.0 - t2881 * t1494 / 8.0;
  t3088 =
    0.0 - t2668 * rp + t7 * t3082 / 2.0 - t2670 / 2.0 + t97 * t2652 * rp;
  t3091 = t17 * t29 * t3088 - t3013 + t3015 + t3017 + t3019;
  t3180 =
    -4.0 * t138 * t2637 * t2 * t2937 + t85 * t3071 * t1302 +
    (i) * t2045 * t3075 + t1316 * t795 * t3091 +
    t795 * t2684 / 2.0 - t97 * t2685 * rp + t90 * t87 * (0.0 - t2939 +
							 t2948 +
							 t2963 +
							 t2970 +
							 t17 *
							 t3 *
							 t3088 +
							 t17 *
							 t29 *
							 (-4.0 *
							  t138 *
							  t2644 *
							  t2 *
							  t2937 +
							  t85 *
							  t3082 *
							  t1302 +
							  t2256 *
							  t230 *
							  t2651 *
							  rp -
							  t230 *
							  t2651 *
							  t1302 +
							  4.0 *
							  t877 *
							  t2652 *
							  t2049 +
							  t1606 *
							  t87 *
							  t1803 *
							  t2679 +
							  t2276 *
							  t2677 *
							  t1115 *
							  t2618 *
							  t2057) +
							 t2975 +
							 t2979 +
							 t2953) /
    2.0 + t230 * t2657 * t1302 - 4.0 * t877 * t2658 * t2049 +
    t98 * t3091 * t2 * t1309 - t85 * t2704 * rp +
    t7 * (2.0 * rp * t2701 +
	  t8 * ((i) * t2045 * t3059 + t230 * t2624 * t1302 -
		4.0 * t877 * t2625 * t2049 +
		(1.0 / 4.0 * i) * t138 * a * t1691 * t830 *
		t2639 +
		t1313 * t16 * t2054 * t233 * t2618 * t2057 +
		4.0 * t138 * t2633 * t2 * t2937 -
		t85 * t3065 * t1302) - 2.0 * t142 * t1947 - t2069) / 2.0;
  t3185 = t173 * t403 * t110;
  t3188 = t2440 * t3185 + t2363 * t2426;
  t3189 = t7 * t3188 / 4.0;
  t3193 = t403 * t53 * t56;
  t3194 = t132 * t3193;
  t3196 = 0.0 - t3189 / 2.0 + t2155 * t2236 * t3194;
  t3198 = t8 * t3196 - t2717;
  t3202 = (-1.0 / 2.0 * i) * t11 * t7;
  t3203 = t233 * t403;
  t3204 = t3203 * t110;
  t3207 = 0.0 - t2559 * t3204 - t2507 * t2388;
  t3210 = t1163 * t3193;
  t3212 = t7 * t3207 / 16.0 + t2321 * t3210;
  t3219 = t85 * t3198;
  t3223 = t85 * t3188 / 4.0;
  t3229 = t2448 * t3185 / 2.0 + t2746 * t2426 / 4.0;
  t3237 = t3203 * t822;
  t3239 =
    t3223 * rp - t7 * t3229 / 2.0 + t1606 * t1075 * t3194 +
    (-1.0 / 4.0 * i) * t12 * t16 * t2433 * t3237;
  t3243 = 2.0 * rp * t3196 + t8 * t3239 - 2.0 * t142 * t2481 - t26 * t2725;
  t3248 = t1324 * t3212;
  t3251 = t1324 * t3212 * rp;
  t3253 = t85 * t3207 / 8.0;
  t3259 = 0.0 - t2941 * t3204 / 4.0 - t2881 * t2388 / 8.0;
  t3266 = t1115 * t403;
  t3267 = t3266 * t822;
  t3269 =
    0.0 - t3253 * rp + t7 * t3259 / 2.0 +
    t1701 * t2589 * t1115 * t3193 + t1862 * t2589 * t3267;
  t3283 = t2052 * t2236;
  mgv5 =
    (t7 *
     (t8 * (t240 + t792) - t26 * t931 +
      a * (t90 * t87 *
	   (t17 * t29 * t958 +
	    t961 * nmu * t98 * t132 * t964) / 2.0 -
	   t580 * t21 * t267 / 4.0 +
	   (-1.0 / 4.0 * i) * t95 * nmu * t98 * t978 -
	   t7 * (t8 *
		 (t675 * t171 * t780 + t462 * t186 * t771 +
		  t365 * t208 * t764 + t675 * t148 * t760 +
		  t635 * t988 / 4.0 + t365 * t12 * t634 * t993) -
		 t774 * t174 * t998 / 4.0) / 2.0 +
	   (-1.0 / 4.0 * i) * t145 * t867 * t870 +
	   (1.0 / 4.0 * i) * t167 * t867 * t870 +
	   t1012 * t200 * t870)) / 2.0 +
     3.0 / 2.0 * t145 * t1036 - t167 * t1036 / 2.0 -
     2.0 * t139 * t140 * t1036 - t164 * t165 * t1036 / 2.0 +
     t90 * t87 * t1457 / 2.0 + t1461 * t1462 * t1475 +
     t90 * t92 * t1474 / 4.0 + 3.0 * t95 * t1474 +
     t1484 * t86 * t1475) * DelTH;
  mgv6 =
    (t7 * t1612 / 2.0 + 3.0 / 2.0 * t145 * t1621 -
     t167 * t1621 / 2.0 - 2.0 * t139 * t140 * t1621 -
     t164 * t165 * t1621 / 2.0 + t90 * t87 * t1717 / 2.0 +
     t1461 * t1462 * t1729 + t90 * t92 * t1728 / 4.0 +
     3.0 * t95 * t1728 + t1484 * t86 * t1729) * DelTH1 - (t1929 +
							  t2111) * DelTH;
  mgv4 = mgv5 + mgv6;
  mgv3 =
    mgv4 + (t2114 * t2127 +
	    t90 * t87 * t2142) * DelTH2 / 2.0 -
    2.0 * (t2147 * t2127 * t1302 + t2114 * t2165 / 2.0 +
	   t1316 * t2168 + t98 * t2170 * t1309 +
	   t90 * t87 * t2193 / 2.0) * DelTH1 + t2327 * DelTH;
  mgv4 = DelR;
  mgv2 = mgv3 * mgv4;
  mgv3 =
    ((t7 * t2475 / 2.0 + 3.0 / 2.0 * t145 * t2481 -
      t167 * t2481 / 2.0 - 2.0 * t139 * t2485 -
      t164 * t165 * t2481 / 2.0 + t90 * t2597 / 2.0 +
      t1461 * t2606 + t90 * t92 * t2604 / 4.0 +
      3.0 * t95 * t2604 + t1484 * t2613) * DelTH + (t7 * t2637 +
						    t90 * t2658) *
     DelTH1 / 2.0 - (t2662 * t1302 + t1316 * t2664 +
		     t90 * t2685 / 2.0 +
		     t98 * t2657 * t2 * t1309 +
		     t7 * t2704 / 2.0) * DelTH) * DelR1 -
    ((t2858 + t3052) * DelTH +
     (0.0 - t2662 * rp + t7 * t3071 / 2.0 + t2664 / 2.0 - t97 * t3075 +
      t90 * t87 * t3091 / 2.0) * DelTH1 - t3180 * DelTH) * DelR;
  mgv1 = mgv2 + mgv3;
  mgv2 =
    mgv1 + (t7 * t3198 / 2.0 + t3202 * t1757 * t29 * t3212) * DelTH * DelR2;
  mgv3 = mgv2;
  mgv5 =
    -2.0 * (0.0 - t3219 * rp + t7 * t3243 / 2.0 +
	    (-1.0 / 2.0 * i) * t7 * t87 * t3248 +
	    (i) * t98 * t3251 + t3202 * t1757 * t29 * t3269) * DelTH * DelR1;
  mgv6 =
    (4.0 * t138 * t3198 * t1 - 2.0 * t85 * t3243 * rp - t3219 +
     t7 * (0.0 - t3189 + t3283 * t3194 + 4.0 * rp * t3239 +
	   t8 * (0.0 - t138 * t3188 * t1 + 2.0 * t85 * t3229 * rp +
		 t3223 - t7 * (t670 * t3185 + t2811) / 4.0 +
		 (-1.0 / 2.0 * i) * t16 * a * t2794 * t3237 +
		 (i) * t12 * t127 * t2433 * t3203 * t2801 +
		 t2768 * t3194) + t3283 * t2418 + t2479 -
	   4.0 * t142 * t2725 - t26 * (t2796 + t2803 + t2797 -
				       t2806 + 2.0 * t2808 +
				       t2445 - t7 * (0.0 - t2811 -
						     t2815) /
				       4.0)) / 2.0 +
     (2.0 * i) * t85 * t87 * t3251 +
     (0.0 - i) * t7 * t87 * t1324 * t3269 +
     (-4.0 * i) * t11 * t271 * t1324 * t3212 * t1 +
     (2.0 * i) * t11 * t230 * t1324 * t3269 * rp +
     (i) * t98 * t3248 +
     t3202 * t1757 * t29 * (t138 * t3207 * t1 / 2.0 -
			    2.0 * t85 * t3259 * rp - t3253 +
			    t7 * (0.0 - t330 * t3204 - t2622) / 8.0 +
			    (-1.0 / 4.0 * i) * t16 * t2589 *
			    t3267 +
			    t2964 * t2589 * t3266 * t2801 +
			    t1861 * t342 * a * t3210)) * DelTH * DelR;
  mgv4 = mgv5 + mgv6;
  t3364 = mgv3 + mgv4;

//    Maple generated expression ends here

  delta = r * r + a * a - 2.0 * mass * r;
  cs2 = (r * r + a * a) * (r * r + a * a) - a * a * delta * stheta * stheta;
  rho = -1.0 / (r - a * i * ctheta);
  TT = (r * r + a * a * ctheta * ctheta) * t3364 / (rho * rho * rho * rho);

  TT = TT * (cos (-mm * a / (rpl - rm) * log ((r - rpl) / (r - rm))) +
	     i * sin (-mm * a / (rpl - rm) * log ((r - rpl) / (r - rm))));
  TT = TT * pow(1.0 - exp(-(timer-start)/25.0),4);
  //TT = TT / ( 1.0 + exp(-(timer-start-100.0)/25.0) ); 

  tred[j + it - PTSY / 2][k + ip - PTSX / 2] =
    (8.0 * pie * delta * TT.real / cs2 / (r * r * r));
  timd[j + it - PTSY / 2][k + ip - PTSX / 2] =
    (8.0 * pie * delta * TT.img / cs2 / (r * r * r));

}

}

/* -------------------------------------------------- */


